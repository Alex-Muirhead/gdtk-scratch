#include "hip/hip_runtime.h"
// simulate.cu
// Include file for chicken, high-level simulation functions.
//
// PJ 2022-09-09

#ifndef SIMULATE_INCLUDED
#define SIMULATE_INCLUDED

#include <cmath>
#include <cstdio>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>
#include <limits>
#include <chrono>
#include <omp.h>

#include "number.cu"
#include "vector3.cu"
#include "gas.cu"
#include "flow.cu"
#include "vertex.cu"
#include "face.cu"
#include "cell.cu"
#include "block.cu"
#include "config.cu"

using namespace std;

namespace SimState {
    number dt = 0.0;
    int step = 0;
    number t = 0.0;
    number t_plot = 0.0;
    int next_plot_indx = 1;
    int steps_since_last_plot = 0;
};


vector<Block> fluidBlocks;
Block* fluidBlocks_on_gpu;
vector<BConfig> blk_configs;
BConfig* blk_configs_on_gpu;
FlowState* flowStates_on_gpu;

// The exchange boundary condition needs to see the collections of data,
// so include the boundary-condition functions after defining those collections.
#include "bcs.cu"

//-------------------------------------------------------------------------------------------------

__host__
void configure_exchange_info(vector<Block>& blks, vector<BConfig>& cfgs)
// Set up the per-face information for the exchange boundary condition.
// Do this after all blocks have been configured wecause we need to dip into
// the other block to get the cell indices on the corresponding boundary face.
{
    for (int iblk=0; iblk < Config::nFluidBlocks; ++iblk) {
        BConfig& cfg = cfgs[iblk];
        if (!cfg.active) continue;
        Block& blk = blks[iblk];
        if (cfg.bcCodes[Face::iminus] == BCCode::exchange) { // jk across face
            int other_i = cfg.i - 1;
            if (other_i < 0) { other_i = Config::nib-1; } // Wrap around.
            int other_j = cfg.j;
            int other_k = cfg.k;
            int other_id = Config::blk_ids[other_i][other_j][other_k];
            Block& other_blk = blks[other_id];
            BConfig& other_cfg = cfgs[other_id];
            for (int k=0; k < cfg.nkc; k++) {
                for (int j=0; j < cfg.njc; j++) {
                    FVFace& f = blk.faces[cfg.iFaceIndex(0, j, k)];
                    FVFace& other_f = other_blk.faces[other_cfg.iFaceIndex(other_cfg.nic, j, k)];
                    f.other_blkId = other_id;
                    f.other_cells[0] = other_f.left_cells[0];
                    f.other_cells[1] = other_f.left_cells[1];
                } // end for j
            } // end for k
        }
        if (cfg.bcCodes[Face::iplus] == BCCode::exchange) { // jk across face
            int other_i = cfg.i + 1;
            if (other_i >= Config::nib) { other_i = 0; } // Wrap around.
            int other_j = cfg.j;
            int other_k = cfg.k;
            int other_id = Config::blk_ids[other_i][other_j][other_k];
            Block& other_blk = fluidBlocks[other_id];
            BConfig& other_cfg = blk_configs[other_id];
            for (int k=0; k < cfg.nkc; k++) {
                for (int j=0; j < cfg.njc; j++) {
                    FVFace& f = blk.faces[cfg.iFaceIndex(cfg.nic, j, k)];
                    FVFace& other_f = other_blk.faces[other_cfg.iFaceIndex(0, j, k)];
                    f.other_blkId = other_id;
                    f.other_cells[0] = other_f.right_cells[0];
                    f.other_cells[1] = other_f.right_cells[1];
                } // end for j
            } // end for k
        }
        if (cfg.bcCodes[Face::jminus] == BCCode::exchange) { // ik across face
            int other_i = cfg.i;
            int other_j = cfg.j - 1;
            if (other_j < 0) { other_j = Config::njb-1; } // Wrap around.
            int other_k = cfg.k;
            int other_id = Config::blk_ids[other_i][other_j][other_k];
            Block& other_blk = fluidBlocks[other_id];
            BConfig& other_cfg = blk_configs[other_id];
            for (int k=0; k < cfg.nkc; k++) {
                for (int i=0; i < cfg.nic; i++) {
                    FVFace& f = blk.faces[cfg.jFaceIndex(i, 0, k)];
                    FVFace& other_f = other_blk.faces[other_cfg.jFaceIndex(i, other_cfg.njc, k)];
                    f.other_blkId = other_id;
                    f.other_cells[0] = other_f.left_cells[0];
                    f.other_cells[1] = other_f.left_cells[1];
                } // end for i
            } // end for k
        }
        if (cfg.bcCodes[Face::jplus] == BCCode::exchange) { // ik across face
            int other_i = cfg.i;
            int other_j = cfg.j + 1;
            if (other_j >= Config::njb) { other_j = 0; } // Wrap around.
            int other_k = cfg.k;
            int other_id = Config::blk_ids[other_i][other_j][other_k];
            Block& other_blk = fluidBlocks[other_id];
            BConfig& other_cfg = blk_configs[other_id];
            for (int k=0; k < cfg.nkc; k++) {
                for (int i=0; i < cfg.nic; i++) {
                    FVFace& f = blk.faces[cfg.jFaceIndex(i, cfg.njc, k)];
                    FVFace& other_f = other_blk.faces[other_cfg.jFaceIndex(i, 0, k)];
                    f.other_blkId = other_id;
                    f.other_cells[0] = other_f.right_cells[0];
                    f.other_cells[1] = other_f.right_cells[1];
                } // end for i
            } // end for k
        }
        if (cfg.bcCodes[Face::kminus] == BCCode::exchange) { // ij across face
            int other_i = cfg.i;
            int other_j = cfg.j;
            int other_k = cfg.k - 1;
            if (other_k < 0) { other_k = Config::nkb-1; } // Wrap around.
            int other_id = Config::blk_ids[other_i][other_j][other_k];
            Block& other_blk = fluidBlocks[other_id];
            BConfig& other_cfg = blk_configs[other_id];
            for (int j=0; j < cfg.njc; j++) {
                for (int i=0; i < cfg.nic; i++) {
                    FVFace& f = blk.faces[cfg.kFaceIndex(i, j, 0)];
                    FVFace& other_f = other_blk.faces[other_cfg.kFaceIndex(i, j, other_cfg.nkc)];
                    f.other_blkId = other_id;
                    f.other_cells[0] = other_f.left_cells[0];
                    f.other_cells[1] = other_f.left_cells[1];
                } // end for i
            } // end for j
        }
        if (cfg.bcCodes[Face::kplus] == BCCode::exchange) { // ij across face
            int other_i = cfg.i;
            int other_j = cfg.j;
            int other_k = cfg.k + 1;
            if (other_k >= Config::nkb) { other_k = 0; } // Wrap around.
            int other_id = Config::blk_ids[other_i][other_j][other_k];
            Block& other_blk = fluidBlocks[other_id];
            BConfig& other_cfg = blk_configs[other_id];
            for (int j=0; j < cfg.njc; j++) {
                for (int i=0; i < cfg.nic; i++) {
                    FVFace& f = blk.faces[cfg.kFaceIndex(i, j, cfg.nkc)];
                    FVFace& other_f = other_blk.faces[other_cfg.kFaceIndex(i, j, 0)];
                    f.other_blkId = other_id;
                    f.other_cells[0] = other_f.right_cells[0];
                    f.other_cells[1] = other_f.right_cells[1];
                } // end for j
            } // end for k
        }
    } // end for iblk
} // end configure_exchange_info()


__device__
void apply_convective_boundary_condition(FVFace& f, FVCell cells[],
                                         FlowState flowStates[], Block blks[])
// For a given FVFace, set the ghost cell FlowStates according to the type of boundary condition.
// Input:
// f:          reference to the FVFace
// cells:      array of cells in the current block
// flowStates: array of FlowStates needed by inflow boundary condition
// blks:       array of Block objects needed by the exchange boundary condition
//
{
    if (f.bcCode < 0) return; // Interior face, leave now.
    //
    switch (f.bcCode) {
    case BCCode::wall_with_slip:
    case BCCode::wall_no_slip_adiabatic:
    case BCCode::wall_no_slip_fixed_T: {
        // Copy data, reflecting velocity.
        if (f.bcId == Face::iplus || f.bcId == Face::jplus || f.bcId == Face::kplus) {
            FVCell& c = cells[f.left_cells[0]];
            FlowState& fs0 = cells[f.right_cells[0]].fs;
            fs0 = c.fs;
            fs0.vel.transform_to_local_frame(f.n, f.t1, f.t2);
            fs0.vel.x = -(fs0.vel.x);
            fs0.vel.transform_to_global_frame(f.n, f.t1, f.t2);
            FlowState& fs1 = cells[f.right_cells[1]].fs;
            fs1 = c.fs;
            fs1.vel.transform_to_local_frame(f.n, f.t1, f.t2);
            fs1.vel.x = -(fs1.vel.x);
            fs1.vel.transform_to_global_frame(f.n, f.t1, f.t2);
        } else {
            FVCell& c = cells[f.right_cells[0]];
            FlowState& fs0 = cells[f.left_cells[0]].fs;
            fs0 = c.fs;
            fs0.vel.transform_to_local_frame(f.n, f.t1, f.t2);
            fs0.vel.x = -(fs0.vel.x);
            fs0.vel.transform_to_global_frame(f.n, f.t1, f.t2);
            FlowState& fs1 = cells[f.left_cells[1]].fs;
            fs1 = c.fs;
            fs1.vel.transform_to_local_frame(f.n, f.t1, f.t2);
            fs1.vel.x = -(fs1.vel.x);
            fs1.vel.transform_to_global_frame(f.n, f.t1, f.t2);
        }
        break;
    }
    case BCCode::exchange: {
        Block& other_blk = blks[f.other_blkId];
        FVCell* other_cells = other_blk.cells_on_gpu;
        // Note that this function only works from within a kernel.
        // On the CPU, we ould like to do the following but it is not allowed from a __device__ function.
        // FVCell* other_cells = other_blk.cells.data();
        if (f.bcId == Face::iplus || f.bcId == Face::jplus || f.bcId == Face::kplus) {
            cells[f.right_cells[0]].fs = other_cells[f.other_cells[0]].fs;
            cells[f.right_cells[1]].fs = other_cells[f.other_cells[1]].fs;
        } else {
            cells[f.left_cells[0]].fs = other_cells[f.other_cells[0]].fs;
            cells[f.left_cells[1]].fs = other_cells[f.other_cells[1]].fs;
        }
        break;
    }
    case BCCode::inflow: {
        FlowState& inflow = flowStates[f.inflowId];
        if (f.bcId == Face::iplus || f.bcId == Face::jplus || f.bcId == Face::kplus) {
            cells[f.right_cells[0]].fs = inflow;
            cells[f.right_cells[1]].fs = inflow;
        } else {
            cells[f.left_cells[0]].fs = inflow;
            cells[f.left_cells[1]].fs = inflow;
        }
        break;
    }
    case BCCode::outflow: {
        if (f.bcId == Face::iplus || f.bcId == Face::jplus || f.bcId == Face::kplus) {
            FVCell& c = cells[f.left_cells[0]];
            cells[f.right_cells[0]].fs = c.fs;
            cells[f.right_cells[1]].fs = c.fs;
        } else {
            FVCell& c = cells[f.right_cells[0]];
            cells[f.left_cells[0]].fs = c.fs;
            cells[f.left_cells[1]].fs = c.fs;
        }
        break;
    }
    default:
        // Do nothing.
        break;
    }
} // end apply_convective_boundary_condition()

//------------------------------------------------------------------------------------------------

__host__
void initialize_simulation(int tindx_start)
{
    char nameBuf[256];
    filesystem::path pth{Config::job};
    if (!filesystem::exists(pth) || !filesystem::is_directory(pth)) {
        throw runtime_error("Job directory is not present in current directory.");
    }
    auto clock_start = chrono::system_clock::now();
    blk_configs = read_config_file(Config::job + "/config.json");
    // Read initial grids and flow data
    size_t bytes_allocated = 0;
    size_t cells_in_simulation = 0;
    for (int blk_id=0; blk_id < Config::nFluidBlocks; ++blk_id) {
        BConfig& cfg = blk_configs[blk_id];
        bytes_allocated += sizeof(BConfig);
        int i = cfg.i; int j = cfg.j; int k = cfg.k;
        if (blk_id != Config::blk_ids[i][j][k]) {
            throw runtime_error("blk_id=" + to_string(blk_id) + " is inconsistent: i=" +
                                to_string(i) + " j=" + to_string(j) + " k=" + to_string(k));
        }
        cfg.fill_in_dimensions(Config::nics[i], Config::njcs[j], Config::nkcs[k]);
        Block blk;
        bytes_allocated += sizeof(Block) + blk.configure(cfg);
        sprintf(nameBuf, "/grid/grid-%04d-%04d-%04d.gz", i, j, k);
        string fileName = Config::job + string(nameBuf);
        blk.readGrid(cfg, fileName);
        sprintf(nameBuf, "/flow/t%04d/flow-%04d-%04d-%04d.zip", tindx_start, i, j, k);
        fileName = Config::job + string(nameBuf);
        blk.readFlow(cfg, fileName);
        blk.computeGeometry(cfg);
        fluidBlocks.push_back(blk);
        if (cfg.active) cells_in_simulation += cfg.nic*cfg.njc*cfg.nkc;
    }
    cout << "Cells in simulation: " << cells_in_simulation << endl;
    cout << "Bytes allocated on CPU: " << fixed << setprecision(3) << bytes_allocated/1.0e6 << "MB" << endl;
    configure_exchange_info(fluidBlocks, blk_configs);
#ifdef CUDA
    // We need to put a copy of the block and config data onto the GPU.
    int nbytes = blk_configs.size()*sizeof(BConfig);
    auto status = hipMalloc(&blk_configs_on_gpu, nbytes);
    if (status) {
        cerr << hipGetErrorString(hipGetLastError()) << endl;
        throw runtime_error("Could not allocate blk_configs on gpu.");
    }
    status = hipMemcpy(blk_configs_on_gpu, blk_configs.data(), nbytes, hipMemcpyHostToDevice);
    if (status) {
        cerr << hipGetErrorString(hipGetLastError()) << endl;
        throw runtime_error("Could not copy blk_configs to gpu.");
    }
    //
    nbytes = fluidBlocks.size()*sizeof(Block);
    status = hipMalloc(&fluidBlocks_on_gpu, nbytes);
    if (status) {
        cerr << hipGetErrorString(hipGetLastError()) << endl;;
        throw runtime_error("Could not allocate fluidBlocks on gpu.");
    }
    status = hipMemcpy(fluidBlocks_on_gpu, fluidBlocks.data(), nbytes, hipMemcpyHostToDevice);
    if (status) {
        cerr << hipGetErrorString(hipGetLastError()) << endl;;
        throw runtime_error("Could not copy fluidBlocks to gpu.");
    }
#endif
    //
    // Set up the simulation control parameters.
    SimState::t = 0.0;
    // Read times.data file to determine starting time.
    ifstream timesFile(Config::job+"/times.data", ifstream::binary);
    if (timesFile.good()) {
        string line;
        while (getline(timesFile, line)) {
            if (line.empty()) continue;
            if (line.find("#") < string::npos) continue; // Skip the comment.
            stringstream ss(line);
            int tindx; number tme;
            ss >> tindx >> tme;
            if (tindx == tindx_start) {
                SimState::t = tme;
                SimState::next_plot_indx = tindx + 1;
            }
        }
    }
    SimState::t_plot = SimState::t + Config::dt_plot_schedule.get_value(SimState::t);
    SimState::steps_since_last_plot = 0;
    auto clock_now = chrono::system_clock::now();
    auto clock_ms = chrono::duration_cast<chrono::milliseconds>(clock_now - clock_start);
    cout << "initialize_simulation() finished in " << clock_ms.count() << "ms" << endl;
    return;
} // initialize_simulation()


__host__
void write_flow_data(int tindx, number tme)
{
    cout << "Write flow data at tindx=" << tindx
         << " time=" << scientific << setprecision(3) << tme << endl;
    //
    char nameBuf[256];
    sprintf(nameBuf, "%s/flow/t%04d", Config::job.c_str(), tindx);
    string flowDir = string(nameBuf);
    if (!filesystem::exists(flowDir)) { filesystem::create_directories(flowDir); }
    for (int blk_id=0; blk_id < Config::nFluidBlocks; ++blk_id) {
        BConfig& blk_config = blk_configs[blk_id];
        int i = blk_config.i; int j = blk_config.j; int k = blk_config.k;
        sprintf(nameBuf, "%s/flow-%04d-%04d-%04d.zip", flowDir.c_str(), i, j, k);
        string fileName = string(nameBuf);
        fluidBlocks[blk_id].writeFlow(blk_config, fileName);
    }
    // Update the times file.
    ofstream timesFile(Config::job+"/times.data", ofstream::binary|ofstream::app);
    timesFile << tindx << " " << tme << endl;
    return;
} // end write_flow_data()


__host__
void march_in_time_using_cpu_only()
// Variant of the main simulation function which uses only the CPU.
// We retain this function as a reasonably-easy-to-read reference code,
// while be build the GPU variant.
{
    if (Config::verbosity > 0) cout << "march_in_time_using_cpu_only() start" << endl;
    auto clock_start = chrono::system_clock::now();
    SimState::dt = Config::dt_init;
    SimState::step = 0;
    // A couple of global arrays to regulate the simulation.
    vector<number> allowed_dts; allowed_dts.resize(Config::nFluidBlocks);
    vector<int> bad_cell_counts; bad_cell_counts.resize(Config::nFluidBlocks);
    //
    #pragma omp parallel for
    for (int ib=0; ib < Config::nFluidBlocks; ib++) {
        BConfig& cfg = blk_configs[ib];
        if (cfg.active) {
            fluidBlocks[ib].encodeConserved(cfg, 0);
            if (Config::viscous) fluidBlocks[ib].setup_LSQ_arrays();
        }
    }
    //
    while (SimState::step < Config::max_step && SimState::t < Config::max_time) {
        //
        // Occasionally determine allowable time step.
        if (SimState::step > 0 && (SimState::step % Config::cfl_count)==0) {
            number smallest_dt = numeric_limits<number>::max();
            for (auto& adt : allowed_dts) adt = smallest_dt;
            number cfl = Config::cfl_schedule.get_value(SimState::t);
            #pragma omp parallel for
            for (int ib=0; ib < Config::nFluidBlocks; ib++) {
                BConfig& cfg = blk_configs[ib];
                Block& blk = fluidBlocks[ib];
                if (cfg.active) allowed_dts[ib] = blk.estimate_allowed_dt(cfg, cfl);
            }
            for (auto adt : allowed_dts) smallest_dt = fmin(smallest_dt, adt);
            SimState::dt = smallest_dt;
        }
        //
        // Gas-dynamic update over three stages with TVD-RK3 weights.
        int bad_cell_count = 0;
        // Stage 1.
        // number t = SimState::t; // Only needed if we have time-dependent source terms or BCs.
        apply_boundary_conditions_for_convective_fluxes();
        for (auto& bcc : bad_cell_counts) bcc = 0;
        #pragma omp parallel for
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (cfg.active) {
                blk.calculate_convective_fluxes(Config::flux_calc, Config::x_order);
                if (Config::viscous) {
                    blk.apply_viscous_boundary_conditions();
                    blk.add_viscous_flux();
                }
                bad_cell_counts[ib] = blk.update_stage_1(cfg, SimState::dt);
            }
        }
        for (auto bcc : bad_cell_counts) bad_cell_count += bcc;
        if (bad_cell_count > 0) {
            throw runtime_error("Stage 1 bad cell count: "+to_string(bad_cell_count));
        }
        // Stage 2
        // t = SimState::t + 0.5*SimState::dt;
        apply_boundary_conditions_for_convective_fluxes();
        for (auto& bcc : bad_cell_counts) bcc = 0;
        #pragma omp parallel for
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (cfg.active) {
                blk.calculate_convective_fluxes(Config::flux_calc, Config::x_order);
                if (Config::viscous) {
                    blk.apply_viscous_boundary_conditions();
                    blk.add_viscous_flux();
                }
                bad_cell_counts[ib] = blk.update_stage_2(cfg, SimState::dt);
            }
        }
        for (auto bcc : bad_cell_counts) bad_cell_count += bcc;
        if (bad_cell_count > 0) {
            throw runtime_error("Stage 2 bad cell count: "+to_string(bad_cell_count));
        }
        // Stage 3
        // t = SimState::t + SimState::dt;
        apply_boundary_conditions_for_convective_fluxes();
        for (auto& bcc : bad_cell_counts) bcc = 0;
        #pragma omp parallel for
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (cfg.active) {
                blk.calculate_convective_fluxes(Config::flux_calc, Config::x_order);
                if (Config::viscous) {
                    blk.apply_viscous_boundary_conditions();
                    blk.add_viscous_flux();
                }
                bad_cell_counts[ib] = blk.update_stage_3(cfg, SimState::dt);
            }
        }
        for (auto bcc : bad_cell_counts) bad_cell_count += bcc;
        if (bad_cell_count > 0) {
            throw runtime_error("Stage 3 bad cell count: "+to_string(bad_cell_count));
        }
        // After a successful gasdynamic update, copy the conserved data back to level 0.
        #pragma omp parallel for
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (cfg.active) blk.copy_conserved_data(cfg, 1, 0);
        }
        //
        SimState::t += SimState::dt;
        SimState::step += 1;
        SimState::steps_since_last_plot += 1;
        //
        // Occasionally write the current step and time to the console.
        if (SimState::step > 0 && (SimState::step % Config::print_count)==0) {
            auto clock_now = chrono::system_clock::now();
            auto clock_ms = chrono::duration_cast<chrono::milliseconds>(clock_now - clock_start);
            double wall_clock_elapsed = clock_ms.count()/1000.0;
            double wall_clock_per_step = wall_clock_elapsed / SimState::step;
            double WCtFT = (Config::max_time - SimState::t) / SimState::dt * wall_clock_per_step;
            double WCtMS = (Config::max_step - SimState::step) * wall_clock_per_step;
            cout << "Step=" << SimState::step
                 << " t=" << scientific << setprecision(3) << SimState::t
                 << " dt=" << SimState::dt
                 << " cfl=" << fixed << Config::cfl_schedule.get_value(SimState::t)
                 << " WC=" << wall_clock_elapsed << "s"
                 << " WCtFT=" << WCtFT << "s"
                 << " WCtMS=" << WCtMS << "s"
                 << endl;
        }
        //
        // Occasionally dump the flow data for making plots.
        if (SimState::t >= SimState::t_plot) {
            write_flow_data(SimState::next_plot_indx, SimState::t);
            SimState::steps_since_last_plot = 0;
            SimState::next_plot_indx += 1;
            SimState::t_plot = SimState::t + Config::dt_plot_schedule.get_value(SimState::t);
        }
    } // end while loop
    if (Config::verbosity > 0) cout << "march_in_time_using_cpu_only() end" << endl;
    return;
} // end march_in_time_using_cpu_only()


__host__
void march_in_time_using_gpu()
// Variant of the main simulation function where we may offload work to the GPU.
{
    if (Config::verbosity > 0) cout << "march_in_time_using_gpu() start" << endl;
    auto clock_start = chrono::system_clock::now();
    SimState::dt = Config::dt_init;
    SimState::step = 0;
    //
    // A couple of global variables for keeping an eye on the simulation process.
    int bad_cell_count = 0;
    int* bad_cell_count_on_gpu;
    auto status = hipMalloc(&bad_cell_count_on_gpu, sizeof(int));
    if (status) throw runtime_error("Could not allocate bad_cell_count_on_gpu.");
    //
    int failed_lsq_setup = 0;
    int* failed_lsq_setup_on_gpu;
    status = hipMalloc(&failed_lsq_setup_on_gpu, sizeof(int));
    if (status) throw runtime_error("Could not allocate failed_lsq_setup_on_gpu.");
    status = hipMemcpy(failed_lsq_setup_on_gpu, &failed_lsq_setup, sizeof(int), hipMemcpyHostToDevice);
    if (status) throw runtime_error("Stage 0, could not copy failed_lsq_setup to gpu.");
    //
    long long int* smallest_dt_picos_on_gpu;
    status = hipMalloc(&smallest_dt_picos_on_gpu, sizeof(long long int));
    if (status) throw runtime_error("Could not allocate smallest_dt_picos_on_gpu.");
    //
    for (int ib=0; ib < Config::nFluidBlocks; ib++) {
        BConfig& cfg = blk_configs[ib];
        if (!cfg.active) continue;
        auto& blk = fluidBlocks[ib];
        // Transfer block data, including the initial flow states, to the GPU and encode.
        int nbytes = blk.cells.size()*sizeof(FVCell);
        auto status = hipMemcpy(blk.cells_on_gpu, blk.cells.data(), nbytes, hipMemcpyHostToDevice);
        if (status) {
            cerr << hipGetErrorString(hipGetLastError()) << endl;
            throw runtime_error("Could not copy blk.cells to gpu.");
        }
        // No need to send conserved quantities and their time-derivatives
        // but we do want to send faces and vertices.
        nbytes = blk.faces.size()*sizeof(FVFace);
        status = hipMemcpy(blk.faces_on_gpu, blk.faces.data(), nbytes, hipMemcpyHostToDevice);
        if (status) {
            cerr << hipGetErrorString(hipGetLastError()) << endl;
            throw runtime_error("Could not copy blk.faces to gpu.");
        }
        nbytes = blk.vertices.size()*sizeof(Vector3);
        status = hipMemcpy(blk.vertices_on_gpu, blk.vertices.data(), nbytes, hipMemcpyHostToDevice);
        if (status) {
            cerr << hipGetErrorString(hipGetLastError()) << endl;
            throw runtime_error("Could not copy blk.vertices to gpu.");
        }
        // Now, do the encode of flow states to conserved quantities.
        Block& blk_on_gpu = fluidBlocks_on_gpu[ib];
        BConfig& cfg_on_gpu = blk_configs_on_gpu[ib];
        int nGPUblocks = cfg.nGPUblocks_for_cells;
        int nGPUthreads = cfg.threads_per_GPUblock;
        encodeConserved_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu, 0);
        auto hipError_t = hipGetLastError();
        if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
        if (Config::viscous) {
            nGPUblocks = cfg.nGPUblocks_for_faces;
            nGPUthreads = cfg.threads_per_GPUblock;
            setup_LSQ_arrays_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu, failed_lsq_setup_on_gpu);
            auto hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
        }
    }
    status = hipMemcpy(&failed_lsq_setup, failed_lsq_setup_on_gpu, sizeof(int), hipMemcpyDeviceToHost);
    if (status) throw runtime_error("Could not copy failed_lsq_setup from gpu to host cpu.");
    if (failed_lsq_setup > 0) {
        throw runtime_error("failed_lsq_setup: "+to_string(failed_lsq_setup));
    }
    //
    // Main stepping loop.
    //
    while (SimState::step < Config::max_step && SimState::t < Config::max_time) {
        //
        // Occasionally determine allowable time step.
        if (SimState::step > 0 && (SimState::step % Config::cfl_count)==0) {
            long long int smallest_dt_picos = numeric_limits<long long int>::max();
            status = hipMemcpy(smallest_dt_picos_on_gpu, &smallest_dt_picos,
                sizeof(long long int), hipMemcpyHostToDevice);
            if (status) throw runtime_error("Stage 0, could not copy smallest_dt_picos to gpu.");
            number cfl = Config::cfl_schedule.get_value(SimState::t);
            for (int ib=0; ib < Config::nFluidBlocks; ib++) {
                BConfig& cfg = blk_configs[ib];
                if (!cfg.active) continue;
                Block& blk = fluidBlocks[ib];
                Block& blk_on_gpu = fluidBlocks_on_gpu[ib];
                BConfig& cfg_on_gpu = blk_configs_on_gpu[ib];
                int nGPUblocks = cfg.nGPUblocks_for_cells;
                int nGPUthreads = cfg.threads_per_GPUblock;
                estimate_allowed_dt_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                                       cfl, smallest_dt_picos_on_gpu);
                auto hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            }
            status = hipMemcpy(&smallest_dt_picos, smallest_dt_picos_on_gpu,
                sizeof(long long int), hipMemcpyDeviceToHost);
            if (status) throw runtime_error("Stage 0, could not copy smallest_dt_picos from gpu to host cpu.");
            SimState::dt = smallest_dt_picos * 1.0e-12;
        }
        //
        // Gas-dynamic update over three stages with TVD-RK3 weights.
        bad_cell_count = 0;
        status = hipMemcpy(bad_cell_count_on_gpu, &bad_cell_count, sizeof(int), hipMemcpyHostToDevice);
        if (status) throw runtime_error("Stage 0, could not copy bad_cell_count to gpu.");
        //
        // Stage 1.
        // number t = SimState::t; // Only needed if we have time-dependent source terms or BCs.
        apply_boundary_conditions_for_convective_fluxes();
        // Boundary-conditions are done on the host CPU, affecting only the ghost-cell data,
        // so we copy just the ghost cell data onto the GPU,
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            FVCell* addr_on_cpu = blk.cells.data() + cfg.nActiveCells;
            FVCell* addr_on_gpu = blk.cells_on_gpu + cfg.nActiveCells;
            int nbytes = cfg.nTotalGhostCells*sizeof(FVCell);
            auto status = hipMemcpy(addr_on_gpu, addr_on_cpu, nbytes, hipMemcpyHostToDevice);
            if (status) {
                cerr << hipGetErrorString(hipGetLastError()) << endl;
                throw runtime_error("Stage 1, could not copy ghost cells to gpu.");
            }
        }
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            // Do the stage-1 update on the GPU.
            Block& blk_on_gpu = fluidBlocks_on_gpu[ib];
            BConfig& cfg_on_gpu = blk_configs_on_gpu[ib];
            //
            int nGPUblocks = cfg.nGPUblocks_for_faces;
            int nGPUthreads = cfg.threads_per_GPUblock;
            calculate_convective_fluxes_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                                           Config::flux_calc, Config::x_order);
            auto hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            if (Config::viscous) {
                apply_viscous_boundary_conditions_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu);
                hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
                add_viscous_flux_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu);
                hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            }
            //
            nGPUblocks = cfg.nGPUblocks_for_cells;
            nGPUthreads = cfg.threads_per_GPUblock;
            update_stage_1_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                              SimState::dt, bad_cell_count_on_gpu);
            hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
        }
        status = hipMemcpy(&bad_cell_count, bad_cell_count_on_gpu, sizeof(int), hipMemcpyDeviceToHost);
        if (status) throw runtime_error("Stage 1, could not copy bad_cell_count from gpu to host cpu.");
        if (bad_cell_count > 0) {
            throw runtime_error("Stage 1, bad cell count: "+to_string(bad_cell_count));
        }
        // Copy cell data back to the CPU for just the active cells.
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            int nbytes = cfg.nActiveCells*sizeof(FVCell);
            auto status = hipMemcpy(blk.cells.data(), blk.cells_on_gpu, nbytes, hipMemcpyDeviceToHost);
            if (status) {
                cerr << hipGetErrorString(hipGetLastError()) << endl;
                throw runtime_error("Stage 1, could not copy blk.cells from gpu to cpu.");
            }
        }
        //
        // Stage 2
        // t = SimState::t + 0.5*SimState::dt;
        apply_boundary_conditions_for_convective_fluxes();
        //
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            FVCell* addr_on_cpu = blk.cells.data() + cfg.nActiveCells;
            FVCell* addr_on_gpu = blk.cells_on_gpu + cfg.nActiveCells;
            int nbytes = cfg.nTotalGhostCells*sizeof(FVCell);
            auto status = hipMemcpy(addr_on_gpu, addr_on_cpu, nbytes, hipMemcpyHostToDevice);
            if (status) {
                cerr << hipGetErrorString(hipGetLastError()) << endl;
                throw runtime_error("Stage 2, could not copy ghost cells to gpu.");
            }
        }
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            // Do the stage-1 update on the GPU.
            Block& blk_on_gpu = fluidBlocks_on_gpu[ib];
            BConfig& cfg_on_gpu = blk_configs_on_gpu[ib];
            //
            int nGPUblocks = cfg.nGPUblocks_for_faces;
            int nGPUthreads = cfg.threads_per_GPUblock;
            calculate_convective_fluxes_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                                           Config::flux_calc, Config::x_order);
            auto hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            if (Config::viscous) {
                apply_viscous_boundary_conditions_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu);
                hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
                add_viscous_flux_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu);
                hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            }
            //
            nGPUblocks = cfg.nGPUblocks_for_cells;
            nGPUthreads = cfg.threads_per_GPUblock;
            update_stage_2_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                              SimState::dt, bad_cell_count_on_gpu);
            hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
        }
        status = hipMemcpy(&bad_cell_count, bad_cell_count_on_gpu, sizeof(int), hipMemcpyDeviceToHost);
        if (status) throw runtime_error("Stage 2, could not copy bad_cell_count from gpu to host cpu.");
        if (bad_cell_count > 0) {
            throw runtime_error("Stage 2, bad cell count: "+to_string(bad_cell_count));
        }
        // Copy cell data back to the CPU for just the active cells.
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            int nbytes = cfg.nActiveCells*sizeof(FVCell);
            auto status = hipMemcpy(blk.cells.data(), blk.cells_on_gpu, nbytes, hipMemcpyDeviceToHost);
            if (status) {
                cerr << hipGetErrorString(hipGetLastError()) << endl;
                throw runtime_error("Stage 2, could not copy blk.cells from gpu to cpu.");
            }
        }
        //
        // Stage 3
        // t = SimState::t + SimState::dt;
        apply_boundary_conditions_for_convective_fluxes();
        //
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            FVCell* addr_on_cpu = blk.cells.data() + cfg.nActiveCells;
            FVCell* addr_on_gpu = blk.cells_on_gpu + cfg.nActiveCells;
            int nbytes = cfg.nTotalGhostCells*sizeof(FVCell);
            auto status = hipMemcpy(addr_on_gpu, addr_on_cpu, nbytes, hipMemcpyHostToDevice);
            if (status) {
                cerr << hipGetErrorString(hipGetLastError()) << endl;
                throw runtime_error("Stage 3, could not copy ghost cells to gpu.");
            }
        }
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            // Do the stage-1 update on the GPU.
            Block& blk_on_gpu = fluidBlocks_on_gpu[ib];
            BConfig& cfg_on_gpu = blk_configs_on_gpu[ib];
            //
            int nGPUblocks = cfg.nGPUblocks_for_faces;
            int nGPUthreads = cfg.threads_per_GPUblock;
            calculate_convective_fluxes_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                                           Config::flux_calc, Config::x_order);
            auto hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            if (Config::viscous) {
                apply_viscous_boundary_conditions_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu);
                hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
                add_viscous_flux_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu);
                hipError_t = hipGetLastError();
                if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
            }
            //
            nGPUblocks = cfg.nGPUblocks_for_cells;
            nGPUthreads = cfg.threads_per_GPUblock;
            update_stage_3_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu,
                                                              SimState::dt, bad_cell_count_on_gpu);
            hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
        }
        status = hipMemcpy(&bad_cell_count, bad_cell_count_on_gpu, sizeof(int), hipMemcpyDeviceToHost);
        if (status) throw runtime_error("Stage 3, could not copy bad_cell_count from gpu to host cpu.");
        if (bad_cell_count > 0) {
            throw runtime_error("Stage 3, bad cell count: "+to_string(bad_cell_count));
        }
        // Copy cell data back to the CPU for just the active cells.
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            int nbytes = cfg.nActiveCells*sizeof(FVCell);
            auto status = hipMemcpy(blk.cells.data(), blk.cells_on_gpu, nbytes, hipMemcpyDeviceToHost);
            if (status) {
                cerr << hipGetErrorString(hipGetLastError()) << endl;
                throw runtime_error("Stage 3, could not copy blk.cells from gpu to cpu.");
            }
        }
        // After a successful gasdynamic update, copy the conserved data back to level 0 on the GPU.
        for (int ib=0; ib < Config::nFluidBlocks; ib++) {
            BConfig& cfg = blk_configs[ib];
            Block& blk = fluidBlocks[ib];
            if (!cfg.active) continue;
            Block& blk_on_gpu = fluidBlocks_on_gpu[ib];
            BConfig& cfg_on_gpu = blk_configs_on_gpu[ib];
            int nGPUblocks = cfg.nGPUblocks_for_cells;
            int nGPUthreads = cfg.threads_per_GPUblock;
            copy_conserved_data_on_gpu<<<nGPUblocks,nGPUthreads>>>(blk_on_gpu, cfg_on_gpu, 1, 0);
            auto hipError_t = hipGetLastError();
            if (hipError_t) throw runtime_error(hipGetErrorString(hipError_t));
        }
        //
        SimState::t += SimState::dt;
        SimState::step += 1;
        SimState::steps_since_last_plot += 1;
        //
        // Occasionally write the current step and time to the console.
        if (SimState::step > 0 && (SimState::step % Config::print_count)==0) {
            auto clock_now = chrono::system_clock::now();
            auto clock_ms = chrono::duration_cast<chrono::milliseconds>(clock_now - clock_start);
            double wall_clock_elapsed = clock_ms.count()/1000.0;
            double wall_clock_per_step = wall_clock_elapsed / SimState::step;
            double WCtFT = (Config::max_time - SimState::t) / SimState::dt * wall_clock_per_step;
            double WCtMS = (Config::max_step - SimState::step) * wall_clock_per_step;
            cout << "Step=" << SimState::step
                 << " t=" << scientific << setprecision(3) << SimState::t
                 << " dt=" << SimState::dt
                 << " cfl=" << fixed << Config::cfl_schedule.get_value(SimState::t)
                 << " WC=" << wall_clock_elapsed << "s"
                 << " WCtFT=" << WCtFT << "s"
                 << " WCtMS=" << WCtMS << "s"
                 << endl;
        }
        //
        // Occasionally dump the flow data for making plots.
        if (SimState::t >= SimState::t_plot) {
            write_flow_data(SimState::next_plot_indx, SimState::t);
            SimState::steps_since_last_plot = 0;
            SimState::next_plot_indx += 1;
            SimState::t_plot = SimState::t + Config::dt_plot_schedule.get_value(SimState::t);
        }
    } // end while loop
    if (Config::verbosity > 0) cout << "march_in_time_using_gpu() end" << endl;
    return;
} // end march_in_time_using_gpu()


__host__
void finalize_simulation()
{
    if (SimState::steps_since_last_plot > 0) {
        write_flow_data(SimState::next_plot_indx, SimState::t);
    }
    for (Block& blk : fluidBlocks) {
        blk.releaseMemory();
    }
    return;
} // end finalize_simulation()

#endif
