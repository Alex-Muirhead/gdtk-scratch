#include "hip/hip_runtime.h"
// simulate.cu
// Include file for chicken, high-level simulation functions.
//
// PJ 2022-09-09

#ifndef SIMULATE_INCLUDED
#define SIMULATE_INCLUDED

#include <cmath>
#include <cstdio>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>
#include <limits>

#include "number.cu"
#include "vector3.cu"
#include "gas.cu"
#include "flow.cu"
#include "vertex.cu"
#include "face.cu"
#include "cell.cu"
#include "block.cu"
#include "config.cu"

using namespace std;

namespace SimState {
    number dt = 0.0;
    int step = 0;
    number t = 0.0;
    number t_plot = 0.0;
    int next_plot_indx = 1;
};

vector<Block> fluidBlocks;

__host__
void initialize_simulation(int tindx_start)
{
    char nameBuf[256];
    filesystem::path pth{Config::job};
    if (!filesystem::exists(pth) || !filesystem::is_directory(pth)) {
        throw runtime_error("Job directory is not present in current directory.");
    }
    read_config_file(Config::job + "/config.json");
    // Read initial grids and flow data
    for (int k=0; k < Config::nkb; ++k) {
        for (int j=0; j < Config::njb; ++j) {
            for (int i=0; i < Config::nib; ++i) {
                if (Config::blk_ids[i][j][k] >= 0) {
                    // Only defined blocks in the array will have a non-zero id.
                    Block blk;
                    int blk_id = Config::blk_ids[i][j][k];
                    blk.configure(Config::nics[i], Config::njcs[j], Config::nkcs[k]);
                    sprintf(nameBuf, "/grid/grid-%04d-%04d-%04d.gz", i, j, k);
                    string fileName = Config::job + string(nameBuf);
                    blk.readGrid(fileName);
                    sprintf(nameBuf, "/flow/t%04d/flow-%04d-%04d-%04d.zip", tindx_start, i, j, k);
                    fileName = Config::job + string(nameBuf);
                    blk.readFlow(fileName);
                    blk.computeGeometry();
                    blk.encodeConserved(0);
                    fluidBlocks.push_back(blk);
                    if (blk_id+1 != fluidBlocks.size()) {
                        throw runtime_error("Inconsistent blk_id and position in fluidBlocks array.");
                    }
                }
            }
        }
    }
    if (fluidBlocks.size() != Config::nFluidBlocks) {
        throw runtime_error("Inconsistent number of blocks: "+
                            to_string(fluidBlocks.size())+" "+to_string(Config::nFluidBlocks));
    }
    //
    // Set up the simulation control parameters.
    SimState::t = 0.0;
    // Read times.data file to determine starting time.
    ifstream timesFile(Config::job+"/times.data", ifstream::binary);
    if (timesFile.good()) {
        string line;
        while (getline(timesFile, line)) {
            if (line.empty()) continue;
            if (line.find("#") < string::npos) continue; // Skip the comment.
            stringstream ss(line);
            int tindx; number tme;
            ss >> tindx >> tme;
            if (tindx == tindx_start) {
                SimState::t = tme;
                SimState::next_plot_indx = tindx + 1;
            }
        }
    }
    SimState::t_plot = SimState::t + Config::dt_plot;
    return;
} // initialize_simulation()

__host__
void write_flow_data(int tindx, number tme)
{
    cout << "Write flow data at tindx=" << tindx << " time=" << tme << endl;
    //
    char nameBuf[256];
    sprintf(nameBuf, "%s/flow/t%04d", Config::job.c_str(), tindx);
    string flowDir = string(nameBuf);
    if (!filesystem::exists(flowDir)) { filesystem::create_directories(flowDir); }
    for (int k=0; k < Config::nkb; ++k) {
        for (int j=0; j < Config::njb; ++j) {
            for (int i=0; i < Config::nib; ++i) {
                if (Config::blk_ids[i][j][k] >= 0) {
                    // Only defined blocks in the array will have a non-zero id.
                    int blk_id = Config::blk_ids[i][j][k];
                    sprintf(nameBuf, "%s/flow-%04d-%04d-%04d.zip", flowDir.c_str(), i, j, k);
                    string fileName = string(nameBuf);
                    fluidBlocks[blk_id].writeFlow(fileName);
                }
            }
        }
    }
    // Update the times file.
    ofstream timesFile(Config::job+"/times.data", ofstream::binary|ofstream::app);
    timesFile << tindx << " " << tme << endl;
    return;
} // end write_flow_data()

// Repetitive boundary condition code is hidden here.
#include "bcs.cu"

__host__
void apply_boundary_conditions()
// Since the boundary-condition code needs a view of all blocks and
// most of the coperations are switching between code to copy specific data,
// we expect the CPU to apply the boundary conditions more effectively than the GPU.
// Measurements might tell us otherwise.
{
    for (int iblk=0; iblk < Config::nFluidBlocks; iblk++) {
        BConfig& blk_config = Config::blk_configs[iblk];
        for (int ibc=0; ibc < 6; ibc++) {
            switch (blk_config.bcCodes[ibc]) {
            case BCCode::wall_with_slip: bc_wall_with_slip(iblk, ibc); break;
            case BCCode::wall_no_slip: bc_wall_no_slip(iblk, ibc); break;
            case BCCode::exchange: bc_exchange(iblk, ibc); break;
            case BCCode::inflow: bc_inflow(iblk, ibc, Config::flow_states[blk_config.bc_fs[ibc]]); break;
            case BCCode::outflow: bc_outflow(iblk, ibc); break;
            default:
                throw runtime_error("Invalid bcCode: "+to_string(blk_config.bcCodes[ibc]));
            }
        } // end for ibc
    } // end for iblk
} // end apply_boundary_conditions()

__host__
void march_in_time()
{
    cout << "march_in_time() start" << endl;
    SimState::dt = Config::dt_init;
    SimState::step = 0;
    //
    while (SimState::step < Config::max_step && SimState::t < Config::max_time) {
        //
        // Occasionally determine allowable time step.
        if (SimState::step > 0 && (SimState::step % Config::cfl_count)==0) {
            for (Block& blk : fluidBlocks) {
                SimState::dt = fmin(SimState::dt, blk.estimate_allowed_dt(Config::cfl));
            }
        }
        // Attempt a step, stage 1.
        apply_boundary_conditions();
        //
        int bad_cell_count = 0;
        for (Block& blk : fluidBlocks) {
            // DEBUG
            FVFace& f = blk.iFaces[blk.iFaceIndex(0,0,0)];
            cout << "DEBUG-A ghost-cell via face fs=" << blk.cells[f.left_cells[0]].fs.toString() << endl;
            cout << "ghost cell indexing fs=" << blk.cells[blk.ghostCellIndex(Face::iminus,0,0,0)].fs.toString() << endl;
            //
            blk.calculate_fluxes(Config::x_order);
            bad_cell_count += blk.update_stage_1(SimState::dt);
        }
        if (bad_cell_count == 0) {
            // After a successful step, copy the conserved data back to level 0.
            for (Block& blk : fluidBlocks) {
                blk.copy_conserved_data(1, 0);
            }
        } else {
            throw runtime_error("Bad cell count: "+to_string(bad_cell_count));
        }
        //
        SimState::t += SimState::dt;
        SimState::step += 1;
        //
        if (SimState::step > 0 && (SimState::step % Config::print_count)==0) {
            cout << "Step=" << SimState::step << " t=" << SimState::t
                 << " dt=" << SimState::dt << " cfl=" << Config::cfl
                 << endl;
        }
    } // end while loop
    cout << "march_in_time() end" << endl;
    return;
}

__host__
void finalize_simulation()
{
    // Exercise the writing of flow data, even we have done no calculations.
    write_flow_data(SimState::next_plot_indx, SimState::t);
    return;
}

#endif
