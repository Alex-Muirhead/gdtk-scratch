#include "hip/hip_runtime.h"
// simulate.cu
// Include file for chicken, high-level simulation functions.
//
// PJ 2022-09-09

#ifndef SIMULATE_INCLUDED
#define SIMULATE_INCLUDED

#include <cmath>
#include <cstdio>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>
#include <limits>

#include "number.cu"
#include "vector3.cu"
#include "gas.cu"
#include "flow.cu"
#include "vertex.cu"
#include "face.cu"
#include "cell.cu"
#include "block.cu"
#include "config.cu"

using namespace std;

namespace SimState {
    number dt = 0.0;
    int step = 0;
    number t = 0.0;
    number t_plot = 0.0;
    int next_plot_indx = 1;
    int steps_since_last_plot = 0;
};

vector<Block> fluidBlocks;

__host__
void initialize_simulation(int tindx_start)
{
    char nameBuf[256];
    filesystem::path pth{Config::job};
    if (!filesystem::exists(pth) || !filesystem::is_directory(pth)) {
        throw runtime_error("Job directory is not present in current directory.");
    }
    read_config_file(Config::job + "/config.json");
    // Read initial grids and flow data
    for (int blk_id=0; blk_id < Config::nFluidBlocks; ++blk_id) {
        BConfig& blk_config = Config::blk_configs[blk_id];
        int i = blk_config.i; int j = blk_config.j; int k = blk_config.k;
        if (blk_id != Config::blk_ids[i][j][k]) {
            throw runtime_error("blk_id=" + to_string(blk_id) + " is inconsistent: i=" +
                                to_string(i) + " j=" + to_string(j) + " k=" + to_string(k));
        }
        Block blk;
        blk.configure(Config::nics[i], Config::njcs[j], Config::nkcs[k], blk_config.active);
        sprintf(nameBuf, "/grid/grid-%04d-%04d-%04d.gz", i, j, k);
        string fileName = Config::job + string(nameBuf);
        blk.readGrid(fileName);
        sprintf(nameBuf, "/flow/t%04d/flow-%04d-%04d-%04d.zip", tindx_start, i, j, k);
        fileName = Config::job + string(nameBuf);
        blk.readFlow(fileName);
        blk.computeGeometry();
        if (blk_config.active) { blk.encodeConserved(0); }
        fluidBlocks.push_back(blk);
    }
    //
    // Set up the simulation control parameters.
    SimState::t = 0.0;
    // Read times.data file to determine starting time.
    ifstream timesFile(Config::job+"/times.data", ifstream::binary);
    if (timesFile.good()) {
        string line;
        while (getline(timesFile, line)) {
            if (line.empty()) continue;
            if (line.find("#") < string::npos) continue; // Skip the comment.
            stringstream ss(line);
            int tindx; number tme;
            ss >> tindx >> tme;
            if (tindx == tindx_start) {
                SimState::t = tme;
                SimState::next_plot_indx = tindx + 1;
            }
        }
    }
    SimState::t_plot = SimState::t + Config::dt_plot_schedule.get_value(SimState::t);
    SimState::steps_since_last_plot = 0;
    return;
} // initialize_simulation()

__host__
void write_flow_data(int tindx, number tme)
{
    cout << "Write flow data at tindx=" << tindx << " time=" << tme << endl;
    //
    char nameBuf[256];
    sprintf(nameBuf, "%s/flow/t%04d", Config::job.c_str(), tindx);
    string flowDir = string(nameBuf);
    if (!filesystem::exists(flowDir)) { filesystem::create_directories(flowDir); }
    for (int blk_id=0; blk_id < Config::nFluidBlocks; ++blk_id) {
        BConfig& blk_config = Config::blk_configs[blk_id];
        int i = blk_config.i; int j = blk_config.j; int k = blk_config.k;
        sprintf(nameBuf, "%s/flow-%04d-%04d-%04d.zip", flowDir.c_str(), i, j, k);
        string fileName = string(nameBuf);
        fluidBlocks[blk_id].writeFlow(fileName);
    }
    // Update the times file.
    ofstream timesFile(Config::job+"/times.data", ofstream::binary|ofstream::app);
    timesFile << tindx << " " << tme << endl;
    return;
} // end write_flow_data()

// Repetitive boundary condition code is hidden here.
#include "bcs.cu"

__host__
void apply_boundary_conditions()
// Since the boundary-condition code needs a view of all blocks and
// most of the coperations are switching between code to copy specific data,
// we expect the CPU to apply the boundary conditions more effectively than the GPU.
// Measurements might tell us otherwise.
{
    for (int iblk=0; iblk < Config::nFluidBlocks; iblk++) {
        BConfig& blk_config = Config::blk_configs[iblk];
        if (blk_config.active) {
            for (int ibc=0; ibc < 6; ibc++) {
                switch (blk_config.bcCodes[ibc]) {
                case BCCode::wall_with_slip: bc_wall_with_slip(iblk, ibc); break;
                case BCCode::wall_no_slip: bc_wall_no_slip(iblk, ibc); break;
                case BCCode::exchange: bc_exchange(iblk, ibc); break;
                case BCCode::inflow: bc_inflow(iblk, ibc, Config::flow_states[blk_config.bc_fs[ibc]]); break;
                case BCCode::outflow: bc_outflow(iblk, ibc); break;
                default:
                    throw runtime_error("Invalid bcCode: "+to_string(blk_config.bcCodes[ibc]));
                }
            } // end for ibc
        } // end if active
    } // end for iblk
} // end apply_boundary_conditions()

__host__
void march_in_time()
{
    cout << "march_in_time() start" << endl;
    SimState::dt = Config::dt_init;
    SimState::step = 0;
    //
    while (SimState::step < Config::max_step && SimState::t < Config::max_time) {
        //
        // Occasionally determine allowable time step.
        if (SimState::step > 0 && (SimState::step % Config::cfl_count)==0) {
            number smallest_dt = numeric_limits<number>::max();
            number cfl = Config::cfl_schedule.get_value(SimState::t);
            for (Block& blk : fluidBlocks) {
                if (blk.active) { smallest_dt = fmin(smallest_dt, blk.estimate_allowed_dt(cfl)); }
            }
            SimState::dt = smallest_dt;
        }
        // Attempt a step, stage 1.
        apply_boundary_conditions();
        //
        int bad_cell_count = 0;
        for (Block& blk : fluidBlocks) {
            if (blk.active) {
                blk.calculate_fluxes(Config::x_order);
                bad_cell_count += blk.update_stage_1(SimState::dt);
            }
        }
        if (bad_cell_count == 0) {
            // After a successful step, copy the conserved data back to level 0.
            for (Block& blk : fluidBlocks) {
                if (blk.active) { blk.copy_conserved_data(1, 0); }
            }
        } else {
            throw runtime_error("Bad cell count: "+to_string(bad_cell_count));
        }
        //
        SimState::t += SimState::dt;
        SimState::step += 1;
        SimState::steps_since_last_plot += 1;
        //
        if (SimState::step > 0 && (SimState::step % Config::print_count)==0) {
            cout << "Step=" << SimState::step << " t=" << SimState::t
                 << " dt=" << SimState::dt << " cfl=" << Config::cfl_schedule.get_value(SimState::t)
                 << endl;
        }
        //
        // Occasionally dump the flow data for making plots.
        if (SimState::t >= SimState::t_plot) {
            write_flow_data(SimState::next_plot_indx, SimState::t);
            SimState::steps_since_last_plot = 0;
            SimState::next_plot_indx += 1;
            SimState::t_plot = SimState::t + Config::dt_plot_schedule.get_value(SimState::t);
        }
    } // end while loop
    cout << "march_in_time() end" << endl;
    return;
} // end march_in_time()

__host__
void finalize_simulation()
{
    if (SimState::steps_since_last_plot > 0) {
        write_flow_data(SimState::next_plot_indx, SimState::t);
    }
    return;
} // end finalize_simulation()

#endif
