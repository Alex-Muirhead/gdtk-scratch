#include "hip/hip_runtime.h"
// face.cu
// Include file for chicken.
// PJ 2022-09-11

#ifndef FACE_INCLUDED
#define FACE_INCLUDED

#include <string>
#include <sstream>
#include <vector>
#include <array>
#include <cmath>

#include "number.cu"
#include "vector3.cu"
#include "gas.cu"
#include "vertex.cu"
#include "flow.cu"

using namespace std;


namespace BCCode {
    // Boundary condition codes, to decide what to do for the ghost cells.
    // Periodic boundary conditions should just work if we wrap the index in each direction.
    // There's not enough information here to have arbitrary block connections.
    constexpr int wall_with_slip = 0;
    constexpr int wall_no_slip_adiabatic = 1;
    constexpr int wall_no_slip_fixed_T = 2;
    constexpr int exchange = 3;
    constexpr int inflow = 4;
    constexpr int outflow = 5;

    array<string,6> names{"wall_with_slip", "wall_no_slip_adiabatic", "wall_no_slip_fixed_T",
            "exchange", "inflow", "outflow"};
};

int BC_code_from_name(string name)
{
    if (name == "wall_with_slip") return BCCode::wall_with_slip;
    if (name == "wall_no_slip_adiabatic") return BCCode::wall_no_slip_adiabatic;
    if (name == "wall_no_slip_fixed_T") return BCCode::wall_no_slip_fixed_T;
    if (name == "wall_no_slip") return BCCode::wall_no_slip_adiabatic; // alias
    if (name == "exchange") return BCCode::exchange;
    if (name == "inflow") return BCCode::inflow;
    if (name == "outflow") return BCCode::outflow;
    return BCCode::wall_with_slip;
}


// Interpolation functions that will be used in the fluc calculator.

__host__ __device__
number van_albada_limit1(number a, number b)
// A smooth slope limiter.
{
    constexpr number eps = 1.0e-12;
    number s = (a*b + fabs(a*b) + eps)/(a*a + b*b + eps);
    return s;
}

__host__ __device__
void interp_l2r2_scalar(number qL1, number qL0, number qR0, number qR1,
                        number& qL, number& qR)
// Reconstruct values, qL,qR, at the middle interface for a stencil of 4 cell-centred values.
// Assume equal cell widths.
{
    // Set up differences and limiter values.
    number delLminus = (qL0 - qL1);
    number del = (qR0 - qL0);
    number delRplus = (qR1 - qR0);
    number sL = van_albada_limit1(delLminus, del);
    number sR = van_albada_limit1(del, delRplus);
    // The actual high-order reconstruction, possibly limited.
    qL = qL0 + sL * 0.125 * (3.0*del + delLminus);
    qR = qR0 - sR * 0.125 * (delRplus + 3.0*del);
} // end of interp_l2r2_scalar()


// Flux calculations are done in the context of a face on a cell.

constexpr int cloud_ncmax = 2;
constexpr int cloud_nfmax = 8;
constexpr int cloud_nmax = cloud_ncmax + cloud_nfmax;

struct FVFace {
    Vector3 pos; // midpoint position in space
    number area;
    Vector3 n;  // unit normal
    Vector3 t1; // unit tangent 1
    Vector3 t2; // unit tangent 2
    array<number,CQI::n> F; // flux vector for conserved quantities
    // We will keep connections to the pieces composing the face
    // as indices into global arrays.
    array<int,4> vtx{-1,-1,-1,-1};
    array<int,2> left_cells{-1,-1};
    array<int,2> right_cells{-1,-1};
    // To apply boundary conditions at the face, we need to carry some extra information.
    // Not all faces are boundary faces, so we start with dummy values.
    int bcId{-1};
    int bcCode{-1};
    int other_blkId{-1};
    array<int,2> other_cells{-1,-1};
    int inflowId{-1};
    double TWall{300.0};
    // For the gradient calculations that form part of the viscous fluxes
    // we keep lists of faces and cells that form a cloud of points around
    // this face-centre.
    array<int,cloud_ncmax> cells_in_cloud{-1,-1};
    array<int,cloud_nfmax> faces_in_cloud{-1,-1,-1,-1,-1,-1,-1,-1};
    int cloud_nc = 0;
    int cloud_nf = 0;
    // Prepared least-squares solution for cloud of cell- and face-FlowStates.
    array<number,cloud_nmax> wx, wy, wz;
    // Spatial gradients with respect to directions x,y,z.
    number dvxdx, dvxdy, dvxdz;
    number dvydx, dvydy, dvydz;
    number dvzdx, dvzdy, dvzdz;
    number dTdx, dTdy, dTdz;
    // We also need the FlowState at this face-centre.  It will be set during
    // the convective-flux calculation or by the boundary-condition code for a wall.
    FlowState fs;

    string toString() const
    {
        ostringstream repr;
        repr << "FVFace(pos=" << pos.toString() << ", n=" << n.toString()
             << ", t1=" << t1.toString() << ", t2=" << t2.toString() << ", area=" << area;
        repr << ", vtx=["; for (auto i : vtx) repr << i << ","; repr << "]";
        repr << ", left_cells=["; for (auto i : left_cells) repr << i << ","; repr << "]";
        repr << ", right_cells=["; for (auto i : right_cells) repr << i << ","; repr << "]";
        repr << ")";
        return repr.str();
    }


    // Specific convective-flux calculators here...

    __host__ __device__
    void ausmdv(const FlowState& fsL, const FlowState& fsR)
    // Compute the face's flux vector from left and right flow states.
    // Wada and Liou's flux calculator, implemented from details in their AIAA paper,
    // with hints from Ian Johnston.
    // Y. Wada and M. -S. Liou (1994)
    // A flux splitting scheme with high-resolution and robustness for discontinuities.
    // AIAA-94-0083.
    {
        Vector3 velL = Vector3(fsL.vel);
        Vector3 velR = Vector3(fsR.vel);
        velL.transform_to_local_frame(n, t1, t2);
        velR.transform_to_local_frame(n, t1, t2);
        //
        number rhoL = fsL.gas.rho;
        number pL = fsL.gas.p;
        number pLrL = pL/rhoL;
        number velxL = velL.x;
        number velyL = velL.y;
        number velzL = velL.z;
        number uL = fsL.gas.e;
        number aL = fsL.gas.a;
        number keL = 0.5*(velxL*velxL + velyL*velyL + velzL*velzL);
        number HL = uL + pLrL + keL;
        //
        number rhoR = fsR.gas.rho;
        number pR = fsR.gas.p;
        number pRrR = pR/rhoR;
        number velxR = velR.x;
        number velyR = velR.y;
        number velzR = velR.z;
        number uR = fsR.gas.e;
        number aR = fsR.gas.a;
        number keR = 0.5*(velxR*velxR + velyR*velyR + velzR*velzR);
        number HR = uR + pR/rhoR + keR;
        //
        // This is the main part of the flux calculator.
        //
        // Weighting parameters (eqn 32) for velocity splitting.
        number alphaL = 2.0*pLrL/(pLrL+pRrR);
        number alphaR = 2.0*pRrR/(pLrL+pRrR);
        // Common sound speed (eqn 33) and Mach numbers.
        number am = fmax(aL, aR);
        number ML = velxL/am;
        number MR = velxR/am;
        // Left state:
        // pressure splitting (eqn 34)
        // and velocity splitting (eqn 30)
        number pLplus, velxLplus;
        number dvelxL = 0.5 * (velxL + fabs(velxL));
        if (fabs(ML) <= 1.0) {
            pLplus = pL*(ML+1.0)*(ML+1.0)*(2.0-ML)*0.25;
            velxLplus = alphaL*((velxL+am)*(velxL+am)/(4.0*am) - dvelxL) + dvelxL;
        } else {
            pLplus = pL * dvelxL / velxL;
            velxLplus = dvelxL;
        }
        // Right state:
        // pressure splitting (eqn 34)
        // and velocity splitting (eqn 31)
        number pRminus, velxRminus;
        number dvelxR = 0.5*(velxR-fabs(velxR));
        if (fabs(MR) <= 1.0) {
            pRminus = pR*(MR-1.0)*(MR-1.0)*(2.0+MR)*0.25;
            velxRminus = alphaR*(-(velxR-am)*(velxR-am)/(4.0*am) - dvelxR) + dvelxR;
        } else {
            pRminus = pR * dvelxR / velxR;
            velxRminus = dvelxR;
        }
        // The mass flux. (eqn 29)
        number massL = velxLplus*rhoL;
        number massR = velxRminus*rhoR;
        number mass_half = massL+massR;
        // Pressure flux (eqn 34)
        number p_half = pLplus + pRminus;
        // Momentum flux: normal direction
        // Compute blending parameter s (eqn 37),
        // the momentum flux for AUSMV (eqn 21) and AUSMD (eqn 21)
        // and blend (eqn 36).
        number dp = pL - pR;
        constexpr number K_SWITCH = 10.0;
        dp = K_SWITCH * fabs(dp) / fmin(pL, pR);
        number s = 0.5 * fmin(1.0, dp);
        number rvel2_AUSMV = massL*velxL + massR*velxR;
        number rvel2_AUSMD = 0.5*(mass_half*(velxL+velxR) - fabs(mass_half)*(velxR-velxL));
        number rvel2_half = (0.5+s)*rvel2_AUSMV + (0.5-s)*rvel2_AUSMD;
        // Assemble components of the flux vector (eqn 36).
        F[CQI::mass] = mass_half;
        number vely = (mass_half >= 0.0) ? velyL : velyR;
        number velz = (mass_half >= 0.0) ? velzL : velzR;
        Vector3 momentum{rvel2_half+p_half, mass_half*vely, mass_half*velz};
        momentum.transform_to_global_frame(n, t1, t2);
        F[CQI::xMom] = momentum.x;
        F[CQI::yMom] = momentum.y;
        F[CQI::zMom] = momentum.z;
        number H = (mass_half >= 0.0) ? HL : HR;
        F[CQI::totEnergy] = mass_half*H;
        // When we introduce species, get the species flux lines from the Dlang code.
        // [TODO] PJ 2022-09-11
        return;
    } // end ausmdv()


    __host__ __device__
    void sbp_asf(FlowState& fsL1, FlowState& fsL0, FlowState& fsR0, FlowState& fsR1)
    // Lachlan's and Christine's Summation-By-Parts Alpha-Split Flux calculation function
    {
        int i;
        array<FlowState,4> stencil{fsL1, fsL0, fsR0, fsR1};
        array<Vector3,4> Vel{fsL1.vel, fsL0.vel, fsR0.vel, fsR1.vel};
        for(i=0;i<4;++i)
        {
            Vel[i].transform_to_local_frame(n, t1, t2);
        }
        //
        number v[10][4] , w[10][4]; 
        for(i=0;i<4;++i)
        {
            v[0][i] = stencil[i].gas.rho;
            w[0][i] = Vel[i].x;
            v[1][i] = Vel[i].x * stencil[i].gas.rho;
            w[1][i] = Vel[i].x;
            v[2][i] = Vel[i].y * stencil[i].gas.rho;
            w[2][i] = Vel[i].x;
            v[3][i] = Vel[i].z * stencil[i].gas.rho;
            w[3][i] = Vel[i].x;
            v[4][i] = stencil[i].gas.e * stencil[i].gas.rho;
            w[4][i] = Vel[i].x;
            v[5][i] = Vel[i].x * Vel[i].x * stencil[i].gas.rho;
            w[5][i] = Vel[i].x;
            v[6][i] = Vel[i].y * Vel[i].y * stencil[i].gas.rho;
            w[6][i] = Vel[i].x;
            v[7][i] = Vel[i].z * Vel[i].z * stencil[i].gas.rho;
            w[7][i] = Vel[i].x;
            v[8][i] = stencil[i].gas.p;
            w[8][i] = Vel[i].x;
            v[9][i] = stencil[i].gas.p;
            w[9][i] = 1;
        }

        number f_c[10];
        number f_e[10];

        for(i=0;i<10;++i)
        {
            f_c[i] = (1.0 / 12.0) * (-v[i][0] * w[i][0] + 7.0 * v[i][1] * w[i][1] + 7.0 * v[i][2] * w[i][2] - v[i][3] * w[i][3]);
            f_e[i] = (1.0 / 12.0) * (-v[i][0] * w[i][2] - v[i][2] * w[i][0] + 8 * v[i][1] * w[i][2] + 8 * v[i][2] * w[i][1] - v[i][1] * w[i][3] - v[i][3] * w[i][1]);
        }
        //
        number alpha_mass = 1.0; number alpha_mom = 0.5; number alpha_ie = 0.5; number alpha_ke = 0.0; number alpha_p = 0.0;
        //
        F[CQI::mass] = alpha_mass * f_c[0] + (1.0 - alpha_mass) * f_e[0];
        number mom_x = alpha_mom * f_c[1] + (1.0 - alpha_mom) * f_e[1] + (alpha_p * f_c[9] + (1.0 - alpha_p) * f_e[9]);
        number mom_y = alpha_mom * f_c[2] + (1.0 - alpha_mom) * f_e[2];
        number mom_z = alpha_mom * f_c[3] + (1.0 - alpha_mom) * f_e[3];
        //
        Vector3 momentum{mom_x, mom_y, mom_z};
        momentum.transform_to_global_frame(n, t1, t2);
        F[CQI::xMom] = momentum.x;
        F[CQI::yMom] = momentum.y;
        F[CQI::zMom] = momentum.z;
        F[CQI::totEnergy] = alpha_ie * f_c[4] + (1.0 - alpha_ie) * f_e[4] + (1.0 / 2.0) * (alpha_ke * f_c[5] + (1.0 - alpha_ke) * f_e[5] + alpha_ke * f_c[6] +
           (1.0 - alpha_ke) * f_e[6] + alpha_ke * f_c[7] + (1.0 - alpha_ke) * f_e[7]) + alpha_p * f_c[8] + (1.0 - alpha_p) * f_e[8];
    } // end sbp_asf()


    // And one generic flux calculation function.

    __host__ __device__
    void calculate_convective_flux(FlowState& fsL1, FlowState& fsL0, FlowState& fsR0, FlowState& fsR1,
                                   int flux_calc, int x_order)
    // Generic convective-flux calculation function.
    {
        if (flux_calc == 0) { // FluxCalc::ausmdv (PJ 2022-10-19 Name cannot be seen.)
            // First-order reconstruction is just a copy from the nearest cell centre.
            FlowState fsL{fsL0};
            FlowState fsR{fsR0};
            if (x_order > 1) {
                // We will interpolate only some GasState properties...
                interp_l2r2_scalar(fsL1.gas.rho, fsL0.gas.rho, fsR0.gas.rho, fsR1.gas.rho, fsL.gas.rho, fsR.gas.rho);
                interp_l2r2_scalar(fsL1.gas.e, fsL0.gas.e, fsR0.gas.e, fsR1.gas.e, fsL.gas.e, fsR.gas.e);
                // and make the rest consistent.
                fsL.gas.update_from_rhoe();
                fsR.gas.update_from_rhoe();
                // Velocity components.
                interp_l2r2_scalar(fsL1.vel.x, fsL0.vel.x, fsR0.vel.x, fsR1.vel.x, fsL.vel.x, fsR.vel.x);
                interp_l2r2_scalar(fsL1.vel.y, fsL0.vel.y, fsR0.vel.y, fsR1.vel.y, fsL.vel.y, fsR.vel.y);
                interp_l2r2_scalar(fsL1.vel.z, fsL0.vel.z, fsR0.vel.z, fsR1.vel.z, fsL.vel.z, fsR.vel.z);
            }
            // Use the reconstructed values near the face in a simple flux calculator.
            ausmdv(fsL, fsR);
            // For later use in gradient calculations for viscous fluxes.
            fs.set_as_average(fsL,fsR);
        } else if (flux_calc == 1) { // FluxCalc::sbp_asf
            sbp_asf(fsL1, fsL0, fsR0, fsR1);
            fs.set_as_average(fsL0,fsR0);
        }
    } // end calculate_convective_flux()

    //------------------------------------------------------------------------------------
    // Methods for viscous fluxes, using the spatial gradients.

    __host__ __device__
    void apply_viscous_boundary_condition()
    // Set the FlowState according to the type of boundary condition.
    // Will overwrite some of the FlowState properties computed earlier
    // in the convective-flux calculation.
    {
        switch (bcCode) {
        case BCCode::wall_no_slip_adiabatic:
            fs.vel.set(0.0, 0.0, 0.0);
            break;
        case BCCode::wall_no_slip_fixed_T:
            fs.vel.set(0.0, 0.0, 0.0);
            fs.gas.T = TWall;
            break;
        default:
            // Do nothing.
            break;
        }
    } // end apply_viscous_boundary_condition()

    __host__ __device__
    void add_viscous_flux()
    // Add the viscous component of the fluxes of mass, momentum and energy
    // to the convective flux values that were computed eariler.
    {
        // Combine the flow-quantity gradients with the transport coefficients.
        number mu, k;
        fs.gas.trans_coeffs(mu, k);
        number lmbda = -2.0/3.0 * mu;
        // Shear stresses.
        number tau_xx = 2.0*mu*dvxdx + lmbda*(dvxdx + dvydy + dvzdz);
        number tau_yy = 2.0*mu*dvydy + lmbda*(dvxdx + dvydy + dvzdz);
        number tau_zz = 2.0*mu*dvzdz + lmbda*(dvxdx + dvydy + dvzdz);
        number tau_xy = mu * (dvxdy + dvydx);
        number tau_xz = mu * (dvxdz + dvzdx);
        number tau_yz = mu * (dvydz + dvzdy);
        // Thermal conduction.
        number qx = k * dTdx;
        number qy = k * dTdy;
        number qz = k * dTdz;
        // Combine into fluxes: store as the dot product (F.n).
        number nx = n.x; number ny = n.y; number nz = n.z;
        // Mass flux -- NO CONTRIBUTION
        F[CQI::xMom] -= tau_xx*nx + tau_xy*ny + tau_xz*nz;
        F[CQI::yMom] -= tau_xy*nx + tau_yy*ny + tau_yz*nz;
        F[CQI::zMom] -= tau_xz*nx + tau_yz*ny + tau_zz*nz;
        F[CQI::totEnergy] -=
            (tau_xx*fs.vel.x + tau_xy*fs.vel.y + tau_xz*fs.vel.z + qx)*nx +
            (tau_xy*fs.vel.x + tau_yy*fs.vel.y + tau_yz*fs.vel.z + qy)*ny +
            (tau_xz*fs.vel.x + tau_yz*fs.vel.y + tau_zz*fs.vel.z + qz)*nz;
    } // end add_viscous_flux()

}; // end FVFace

#endif
