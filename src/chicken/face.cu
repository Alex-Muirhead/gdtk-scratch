#include "hip/hip_runtime.h"
// face.cu
// Include file for chicken.
// PJ 2022-09-11

#ifndef FACE_INCLUDED
#define FACE_INCLUDED

#include <string>
#include <sstream>
#include <vector>
#include <array>

#include "number.cu"
#include "vector3.cu"
#include "vertex.cu"
#include "flow.cu"
#include "gas.cu"

using namespace std;

struct FVFace {
    Vector3 pos; // midpoint position in space
    number area;
    Vector3 n;  // unit normal
    Vector3 t1; // unit tangent 1
    Vector3 t2; // unit tangent 2
    array<number,CQI::n> F; // flux vector for conserved quantities
    // We will keep connections to the pieces composing the face
    // as indices into global arrays.
    array<int,4> vtx{0, 0, 0, 0};
    array<int,2> left_cells{0, 0};
    array<int,2> right_cells{0, 0};

    string toString() {
        ostringstream repr;
        repr << "FVFace(pos=" << pos.toString() << ", n=" << n.toString()
             << ", t1=" << t1.toString() << ", t2=" << t2.toString() << ", area=" << area;
        repr << ", vtx=["; for (auto i : vtx) repr << i << ","; repr << "]";
        repr << ", left_cells=["; for (auto i : left_cells) repr << i << ","; repr << "]";
        repr << ", right_cells=["; for (auto i : right_cells) repr << i << ","; repr << "]";
        repr << ")";
        return repr.str();
    }

    // Specific flux calculators here...

    __host__ __device__
    void ausmdv(const FlowState& fsL, const FlowState& fsR)
    // Compute the face's flux vector from left and right flow states.
    // Wada and Liou's flux calculator, implemented from details in their AIAA paper,
    // with hints from Ian Johnston.
    // Y. Wada and M. -S. Liou (1994)
    // A flux splitting scheme with high-resolution and robustness for discontinuities.
    // AIAA-94-0083.
    {
        Vector3 velL = Vector3(fsL.vel);
        Vector3 velR = Vector3(fsR.vel);
        velL.transform_to_local_frame(n, t1, t2);
        velR.transform_to_local_frame(n, t1, t2);
        //
        number rhoL = fsL.gas.rho;
        number pL = fsL.gas.p;
        number pLrL = pL/rhoL;
        number velxL = velL.x;
        number velyL = velL.y;
        number velzL = velL.z;
        number uL = fsL.gas.e;
        number aL = fsL.gas.a;
        number keL = 0.5*(velxL*velxL + velyL*velyL + velzL*velzL);
        number HL = uL + pLrL + keL;
        //
        number rhoR = fsR.gas.rho;
        number pR = fsR.gas.p;
        number pRrR = pR/rhoR;
        number velxR = velR.x;
        number velyR = velR.y;
        number velzR = velR.z;
        number uR = fsR.gas.e;
        number aR = fsR.gas.a;
        number keR = 0.5*(velxR*velxR + velyR*velyR + velzR*velzR);
        number HR = uR + pR/rhoR + keR;
        //
        // This is the main part of the flux calculator.
        //
        // Weighting parameters (eqn 32) for velocity splitting.
        number alphaL = 2.0*pLrL/(pLrL+pRrR);
        number alphaR = 2.0*pRrR/(pLrL+pRrR);
        // Common sound speed (eqn 33) and Mach numbers.
        number am = fmax(aL, aR);
        number ML = velxL/am;
        number MR = velxR/am;
        // Left state:
        // pressure splitting (eqn 34)
        // and velocity splitting (eqn 30)
        number pLplus, velxLplus;
        number dvelxL = 0.5 * (velxL + fabs(velxL));
        if (fabs(ML) <= 1.0) {
            pLplus = pL*(ML+1.0)*(ML+1.0)*(2.0-ML)*0.25;
            velxLplus = alphaL*((velxL+am)*(velxL+am)/(4.0*am) - dvelxL) + dvelxL;
        } else {
            pLplus = pL * dvelxL / velxL;
            velxLplus = dvelxL;
        }
        // Right state:
        // pressure splitting (eqn 34)
        // and velocity splitting (eqn 31)
        number pRminus, velxRminus;
        number dvelxR = 0.5*(velxR-fabs(velxR));
        if (fabs(MR) <= 1.0) {
            pRminus = pR*(MR-1.0)*(MR-1.0)*(2.0+MR)*0.25;
            velxRminus = alphaR*(-(velxR-am)*(velxR-am)/(4.0*am) - dvelxR) + dvelxR;
        } else {
            pRminus = pR * dvelxR / velxR;
            velxRminus = dvelxR;
        }
        // The mass flux. (eqn 29)
        number massL = velxLplus*rhoL;
        number massR = velxRminus*rhoR;
        number mass_half = massL+massR;
        // Pressure flux (eqn 34)
        number p_half = pLplus + pRminus;
        // Momentum flux: normal direction
        // Compute blending parameter s (eqn 37),
        // the momentum flux for AUSMV (eqn 21) and AUSMD (eqn 21)
        // and blend (eqn 36).
        number dp = pL - pR;
        constexpr number K_SWITCH = 10.0;
        dp = K_SWITCH * fabs(dp) / fmin(pL, pR);
        number s = 0.5 * fmin(1.0, dp);
        number rvel2_AUSMV = massL*velxL + massR*velxR;
        number rvel2_AUSMD = 0.5*(mass_half*(velxL+velxR) - fabs(mass_half)*(velxR-velxL));
        number rvel2_half = (0.5+s)*rvel2_AUSMV + (0.5-s)*rvel2_AUSMD;
        // Assemble components of the flux vector (eqn 36).
        F[CQI::mass] = mass_half;
        number vely = (mass_half >= 0.0) ? velyL : velyR;
        number velz = (mass_half >= 0.0) ? velzL : velzR;
        Vector3 momentum{rvel2_half+p_half, mass_half*vely, mass_half*velz};
        momentum.transform_to_global_frame(n, t1, t2);
        F[CQI::xMom] = momentum.x;
        F[CQI::yMom] = momentum.y;
        F[CQI::zMom] = momentum.z;
        number H = (mass_half >= 0.0) ? HL : HR;
        F[CQI::totEnergy] = mass_half*H;
        // When we introduce species, get the species flux lines from the Dlang code.
        // [TODO] PJ 2022-09-11
        return;
    } // end ausmdv()

    // And one generic flux calculation function.

    __host__ __device__
    void calculate_flux(FlowState& fsL1, FlowState& fsL0, FlowState& fsR0, FlowState& fsR1,
                        number L1, number L0, number R0, number R1, int x_order)
    // Generic fluc calculation function.
    {
        // [TODO] Implement reconstruction and allow other flux calculators.
        ausmdv(fsL0, fsR0);
    }

}; // end FVFace

#endif
