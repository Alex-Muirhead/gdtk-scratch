#include "hip/hip_runtime.h"
// block.cu
// Include file for chicken.
// PJ 2022-09-11

#ifndef BLOCK_INCLUDED
#define BLOCK_INCLUDED

#include <string>
#include <fstream>
#include <sstream>
#include <stdexcept>
#include <limits>
#include "include/bxzstr/bxzstr.hpp"
#include <zip.h>

#include "number.cu"
#include "rsla.cu"
#include "vector3.cu"
#include "config.cu"
#include "gas.cu"
#include "vertex.cu"
#include "flow.cu"
#include "face.cu"
#include "cell.cu"

using namespace std;


__host__ __device__
int setup_LSQ_arrays_at_face(FVFace& f, FVCell cells[], FVFace faces[])
// Prepare the inverse of the least-squares design matrix and use it to
// prepare the final weights of the points in the cloud at each face.
// The evaluation of each flow gradient becomes a matrix*vector product.
//
// Returns 0 if all successful, else 1 if a singular matrix is encountered.
//
// Adapted from the Eilmer4 code PJ, 2022-10-22.
// Placed in block.cu so that FVCell and FVFace struct definitions can be seen.
{
    // Get pointers to all of the cloud positions.
    Vector3* cloud_pos[cloud_nmax];
    for (int i=0; i < f.cloud_nc; i++) { cloud_pos[i] = &(cells[f.cells_in_cloud[i]].pos); }
    for (int i=0; i < f.cloud_nf; i++) { cloud_pos[f.cloud_nc+i] = &(faces[f.faces_in_cloud[i]].pos); }
    int cloud_n = f.cloud_nc + f.cloud_nf;
    //
    // Calculate the weights used in the least-squares gradient calculation.
    // These are the square of the weights on the original linear constraint eqns
    // and are calculated with the face centre as the reference point.
    number weights2[cloud_nmax];
    number x0 = f.pos.x; number y0 = f.pos.y; number z0 = f.pos.z;
    for (int i=0; i < cloud_n; i++) {
        number dx = cloud_pos[i]->x - x0;
        number dy = cloud_pos[i]->y - y0;
        number dz = cloud_pos[i]->z - z0;
        weights2[i] = 1.0/(dx*dx+dy*dy+dz*dz);
    }
    //
    // Set up the matrix for the normal equations.
    //
    number dx[cloud_nmax], dy[cloud_nmax], dz[cloud_nmax];
    number xx = 0.0; number xy = 0.0; number xz = 0.0;
    number yy = 0.0; number yz = 0.0; number zz = 0.0;
    for (int i=0; i < cloud_n; i++) {
        dx[i] = cloud_pos[i]->x - x0;
        dy[i] = cloud_pos[i]->y - y0;
        dz[i] = cloud_pos[i]->z - z0;
        xx += weights2[i]*dx[i]*dx[i];
        xy += weights2[i]*dx[i]*dy[i];
        xz += weights2[i]*dx[i]*dz[i];
        yy += weights2[i]*dy[i]*dy[i];
        yz += weights2[i]*dy[i]*dz[i];
        zz += weights2[i]*dz[i]*dz[i];
    }
    number xTx[3][3]; // normal matrix
    xTx[0][0] = xx; xTx[0][1] = xy; xTx[0][2] = xz;
    xTx[1][0] = xy; xTx[1][1] = yy; xTx[1][2] = yz;
    xTx[2][0] = xz; xTx[2][1] = yz; xTx[2][2] = zz;
    //
    number xTxInv[3][3]; // Inverse of normal matrix.
    number very_small_value = 1.0e-16; // Should be 1.0e-32 (normInf(xTx))^^3;
    if (0 != MInverse(xTx, xTxInv, very_small_value)) {
        return 1;
    }
    // Prepare final weights for later use in the reconstruction phase.
    for (int i=0; i < cloud_n; i++) {
        f.wx[i] = xTxInv[0][0]*dx[i] + xTxInv[0][1]*dy[i] + xTxInv[0][2]*dz[i];
        f.wx[i] *= weights2[i];
        f.wy[i] = xTxInv[1][0]*dx[i] + xTxInv[1][1]*dy[i] + xTxInv[1][2]*dz[i];
        f.wy[i] *= weights2[i];
        f.wz[i] = xTxInv[2][0]*dx[i] + xTxInv[2][1]*dy[i] + xTxInv[2][2]*dz[i];
        f.wz[i] *= weights2[i];
    }
    return 0; // All weights successfully computed.
} // end setup_LSQ_arrays()


__host__ __device__
void calculate_gradients_at_face(FVFace& f, FVCell cells[], FVFace faces[])
// Compute the flow quantity gradients at the face centre,
// making use of the least-squares coefficients prepared at the start of stepping.
{
    // Get pointers to all of the cloud FlowStates.
    FlowState cloud_fs[cloud_nmax];
    for (int i=0; i < f.cloud_nc; i++) { cloud_fs[i] = cells[f.cells_in_cloud[i]].fs; }
    for (int i=0; i < f.cloud_nf; i++) { cloud_fs[f.cloud_nc+i] = faces[f.faces_in_cloud[i]].fs; }
    int cloud_n = f.cloud_nc + f.cloud_nf;
    // Now, compute the gradients, one flow quantity at a time.
    number q0 = f.fs.gas.T;
    f.dTdx = 0.0; f.dTdy = 0.0; f.dTdz = 0.0;
    for (int i=0; i < cloud_n; i++) {
        number dq = cloud_fs[i].gas.T - q0;
        f.dTdx += f.wx[i] * dq;
        f.dTdy += f.wy[i] * dq;
        f.dTdz += f.wz[i] * dq;
    }
    q0 = f.fs.vel.x;
    f.dvxdx = 0.0; f.dvxdy = 0.0; f.dvxdz = 0.0;
    for (int i=0; i < cloud_n; i++) {
        number dq = cloud_fs[i].vel.x - q0;
        f.dvxdx += f.wx[i] * dq;
        f.dvxdy += f.wy[i] * dq;
        f.dvxdz += f.wz[i] * dq;
    }
    q0 = f.fs.vel.y;
    f.dvydx = 0.0; f.dvydy = 0.0; f.dvydz = 0.0;
    for (int i=0; i < cloud_n; i++) {
        number dq = cloud_fs[i].vel.y - q0;
        f.dvydx += f.wx[i] * dq;
        f.dvydy += f.wy[i] * dq;
        f.dvydz += f.wz[i] * dq;
    }
    q0 = f.fs.vel.z;
    f.dvzdx = 0.0; f.dvzdy = 0.0; f.dvzdz = 0.0;
    for (int i=0; i < cloud_n; i++) {
        number dq = cloud_fs[i].vel.z - q0;
        f.dvzdx += f.wx[i] * dq;
        f.dvzdy += f.wy[i] * dq;
        f.dvzdz += f.wz[i] * dq;
    }
} // end calculate_gradients_at_face()

//-----------------------------------------------------------------------------------

struct Block {
    // Storage for active cells and ghost cells.
    vector<FVCell> cells;
    FVCell* cells_on_gpu;
    //
    // Active cells have conserved quantities data, along with the time derivatives.
    vector<ConservedQuantities> Q;
    vector<ConservedQuantities> dQdt;
    ConservedQuantities* Q_on_gpu;
    ConservedQuantities* dQdt_on_gpu;
    //
    // Collection of faces which bound the active cells.
    // We compute fluxes of conserved flow properties across these faces.
    vector<FVFace> faces;
    FVFace* faces_on_gpu;
    //
    // The vertices are used to define the locations and geometric properties
    // of faces and cells.
    vector<Vector3> vertices;
    Vector3* vertices_on_gpu;


    __host__
    string toString() {
        string repr = "Block()";
        return repr;
    }

    __host__
    size_t configure(const BConfig& cfg)
    // Set up the block to hold the grid and flow data.
    // Do this before reading a grid or flow file.
    {
        size_t bytes_allocated = 0;
        // Now that we know the numbers of cells, resize the data store to fit them all.
        cells.resize(cfg.nActiveCells + cfg.nTotalGhostCells);
        bytes_allocated += cells.size()*sizeof(FVCell);
        if (cfg.active) {
            Q.resize(cfg.nActiveCells*2);
            dQdt.resize(cfg.nActiveCells*3);
        }
        bytes_allocated += (Q.size()+dQdt.size())*sizeof(ConservedQuantities);
        //
        // Each set of finite-volume faces is in the index-plane of the corresponding vertices
        // but we pack them all into the one vector.
        faces.resize(cfg.nFaces);
        bytes_allocated += faces.size()*sizeof(FVFace);
        //
        // And the vertices.
        vertices.resize((cfg.nic+1)*(cfg.njc+1)*(cfg.nkc+1));
        bytes_allocated += vertices.size()*sizeof(Vector3);
        //
#ifdef CUDA
        // We need to allocate corresponding memory space on the GPU.
        auto status = hipMalloc(&cells_on_gpu, cells.size()*sizeof(FVCell));
        if (status) throw runtime_error("Could not allocate cells on gpu.");
        status = hipMalloc(&Q_on_gpu, Q.size()*sizeof(ConservedQuantities));
        if (status) throw runtime_error("Could not allocate Q on gpu.");
        status = hipMalloc(&dQdt_on_gpu, dQdt.size()*sizeof(ConservedQuantities));
        if (status) throw runtime_error("Could not allocate dQdt on gpu.");
        status = hipMalloc(&faces_on_gpu, faces.size()*sizeof(FVFace));
        if (status) throw runtime_error("Could not allocate faces on gpu.");
        status = hipMalloc(&vertices_on_gpu, vertices.size()*sizeof(Vector3));
        if (status) throw runtime_error("Could not allocate vertices on gpu.");
#endif
        //
        // Make connections from cells to faces and vertices.
        for (int k=0; k < cfg.nkc; k++) {
            for (int j=0; j < cfg.njc; j++) {
                for (int i=0; i < cfg.nic; i++) {
                    FVCell& c = cells[cfg.activeCellIndex(i,j,k)];
                    c.face[Face::iminus] = cfg.iFaceIndex(i,j,k);
                    c.face[Face::iplus] = cfg.iFaceIndex(i+1,j,k);
                    c.face[Face::jminus] = cfg.jFaceIndex(i,j,k);
                    c.face[Face::jplus] = cfg.jFaceIndex(i,j+1,k);
                    c.face[Face::kminus] = cfg.kFaceIndex(i,j,k);
                    c.face[Face::kplus] = cfg.kFaceIndex(i,j,k+1);
                    c.vtx[0] = cfg.vtxIndex(i,j,k);
                    c.vtx[1] = cfg.vtxIndex(i+1,j,k);
                    c.vtx[2] = cfg.vtxIndex(i+1,j+1,k);
                    c.vtx[3] = cfg.vtxIndex(i,j+1,k);
                    c.vtx[4] = cfg.vtxIndex(i,j,k+1);
                    c.vtx[5] = cfg.vtxIndex(i+1,j,k+1);
                    c.vtx[6] = cfg.vtxIndex(i+1,j+1,k+1);
                    c.vtx[7] = cfg.vtxIndex(i,j+1,k+1);
                }
            }
        }
        //
        // Make connections from faces to cells and vertices.
        //
        // Face i  0     1     2     3     4
        //         +-----+-----+-----+-----+
        // Cell i  |  0  |  1  |  2  |  3  |
        //         +-----+-----+-----+-----+
        //
        // iFaces
        for (int k=0; k < cfg.nkc; k++) {
            for (int j=0; j < cfg.njc; j++) {
                for (int i=0; i < cfg.nic+1; i++) {
                    FVFace& f = faces[cfg.iFaceIndex(i,j,k)];
                    f.vtx[0] = cfg.vtxIndex(i,j,k);
                    f.vtx[1] = cfg.vtxIndex(i,j+1,k);
                    f.vtx[2] = cfg.vtxIndex(i,j+1,k+1);
                    f.vtx[3] = cfg.vtxIndex(i,j,k+1);
                    // Set neighbouring cells for convective fluxes.
                    if (i == 0) {
                        f.left_cells[1] = cfg.ghostCellIndex(Face::iminus,j,k,1);
                        f.left_cells[0] = cfg.ghostCellIndex(Face::iminus,j,k,0);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i+1,j,k);
                        f.bcId = Face::iminus;
                        f.bcCode = cfg.bcCodes[Face::iminus];
                        if (f.bcCode == BCCode::inflow) f.inflowId = cfg.bc_fs[Face::iminus];
                        if (f.bcCode == BCCode::wall_no_slip_fixed_T) f.TWall = cfg.bc_TWall[Face::iminus];
                    } else if (i == 1) {
                        f.left_cells[1] = cfg.ghostCellIndex(Face::iminus,j,k,0);
                        f.left_cells[0] = cfg.activeCellIndex(i-1,j,k);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i+1,j,k);
                    } else if (i == cfg.nic-1) {
                        f.left_cells[1] = cfg.activeCellIndex(i-2,j,k);
                        f.left_cells[0] = cfg.activeCellIndex(i-1,j,k);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.ghostCellIndex(Face::iplus,j,k,0);
                    } else if (i == cfg.nic) {
                        f.left_cells[1] = cfg.activeCellIndex(i-2,j,k);
                        f.left_cells[0] = cfg.activeCellIndex(i-1,j,k);
                        f.right_cells[0] = cfg.ghostCellIndex(Face::iplus,j,k,0);
                        f.right_cells[1] = cfg.ghostCellIndex(Face::iplus,j,k,1);
                        f.bcId = Face::iplus;
                        f.bcCode = cfg.bcCodes[Face::iplus];
                        if (f.bcCode == BCCode::inflow) f.inflowId = cfg.bc_fs[Face::iplus];
                        if (f.bcCode == BCCode::wall_no_slip_fixed_T) f.TWall = cfg.bc_TWall[Face::iplus];
                    } else {
                        // All interior cells.
                        f.left_cells[1] = cfg.activeCellIndex(i-2,j,k);
                        f.left_cells[0] = cfg.activeCellIndex(i-1,j,k);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i+1,j,k);
                    }
                    // Set cloud of FlowStates for gradient calculations of viscous fluxes.
                    if (i == 0) {
                        f.bcCode = cfg.bcCodes[Face::iminus];
                        if (f.bcCode == BCCode::wall_with_slip || f.bcCode == BCCode::wall_with_slip) {
                            // Do not use ghost cell.
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j,k), -1};
                            f.cloud_nc = 1;
                        } else {
                            f.cells_in_cloud = {cfg.ghostCellIndex(Face::iminus,j,k,0), cfg.activeCellIndex(i,j,k)};
                            f.cloud_nc = 2;
                        }
                        f.faces_in_cloud = {cfg.jFaceIndex(i,j,k), cfg.jFaceIndex(i,j+1,k),
                                            cfg.kFaceIndex(i,j,k), cfg.kFaceIndex(i,j,k+1),
                                            -1, -1, -1, -1};
                        f.cloud_nf = 4;
                    } else if (i == cfg.nic) {
                        f.bcCode = cfg.bcCodes[Face::iplus];
                        if (f.bcCode == BCCode::wall_with_slip || f.bcCode == BCCode::wall_with_slip) {
                            // Do not use ghost cell.
                            f.cells_in_cloud = {cfg.activeCellIndex(i-1,j,k), -1};
                            f.cloud_nc = 1;
                        } else {
                            f.cells_in_cloud = {cfg.activeCellIndex(i-1,j,k), cfg.ghostCellIndex(Face::iplus,j,k,0)};
                            f.cloud_nc = 2;
                        }
                        f.faces_in_cloud = {cfg.jFaceIndex(i-1,j,k), cfg.jFaceIndex(i-1,j+1,k),
                                            cfg.kFaceIndex(i-1,j,k), cfg.kFaceIndex(i-1,j,k+1),
                                            -1, -1, -1, -1};
                        f.cloud_nf = 4;
                    } else {
                        f.bcCode = -1; // Interior face.
                        f.cells_in_cloud = {cfg.activeCellIndex(i-1,j,k), cfg.activeCellIndex(i,j,k)};
                        f.cloud_nc = 2;
                        f.faces_in_cloud = {cfg.jFaceIndex(i-1,j,k), cfg.jFaceIndex(i-1,j+1,k),
                                            cfg.kFaceIndex(i-1,j,k), cfg.kFaceIndex(i-1,j,k+1),
                                            cfg.jFaceIndex(i,j,k), cfg.jFaceIndex(i,j+1,k),
                                            cfg.kFaceIndex(i,j,k), cfg.kFaceIndex(i,j,k+1)};
                        f.cloud_nf = 8;
                    }
                }
            }
        }
        // jFaces
        for (int k=0; k < cfg.nkc; k++) {
            for (int i=0; i < cfg.nic; i++) {
                for (int j=0; j < cfg.njc+1; j++) {
                    FVFace& f = faces[cfg.jFaceIndex(i,j,k)];
                    f.vtx[0] = cfg.vtxIndex(i+1,j,k);
                    f.vtx[1] = cfg.vtxIndex(i,j,k);
                    f.vtx[2] = cfg.vtxIndex(i,j,k+1);
                    f.vtx[3] = cfg.vtxIndex(i+1,j,k+1);
                    // Set neighbouring cells for convective fluxes.
                    if (j == 0) {
                        f.left_cells[1] = cfg.ghostCellIndex(Face::jminus,i,k,1);
                        f.left_cells[0] = cfg.ghostCellIndex(Face::jminus,i,k,0);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i,j+1,k);
                        f.bcId = Face::jminus;
                        f.bcCode = cfg.bcCodes[Face::jminus];
                        if (f.bcCode == BCCode::inflow) f.inflowId = cfg.bc_fs[Face::jminus];
                        if (f.bcCode == BCCode::wall_no_slip_fixed_T) f.TWall = cfg.bc_TWall[Face::jminus];
                    } else if (j == 1) {
                        f.left_cells[1] = cfg.ghostCellIndex(Face::jminus,i,k,0);
                        f.left_cells[0] = cfg.activeCellIndex(i,j-1,k);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i,j+1,k);
                    } else if (j == cfg.njc-1) {
                        f.left_cells[1] = cfg.activeCellIndex(i,j-2,k);
                        f.left_cells[0] = cfg.activeCellIndex(i,j-1,k);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.ghostCellIndex(Face::jplus,i,k,0);
                    } else if (j == cfg.njc) {
                        f.left_cells[1] = cfg.activeCellIndex(i,j-2,k);
                        f.left_cells[0] = cfg.activeCellIndex(i,j-1,k);
                        f.right_cells[0] = cfg.ghostCellIndex(Face::jplus,i,k,0);
                        f.right_cells[1] = cfg.ghostCellIndex(Face::jplus,i,k,1);
                        f.bcId = Face::jplus;
                        f.bcCode = cfg.bcCodes[Face::jplus];
                        if (f.bcCode == BCCode::inflow) f.inflowId = cfg.bc_fs[Face::jplus];
                        if (f.bcCode == BCCode::wall_no_slip_fixed_T) f.TWall = cfg.bc_TWall[Face::jplus];
                    } else {
                        // All interior cells.
                        f.left_cells[1] = cfg.activeCellIndex(i,j-2,k);
                        f.left_cells[0] = cfg.activeCellIndex(i,j-1,k);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i,j+1,k);
                    }
                    // Set cloud of FlowStates for gradient calculations of viscous fluxes.
                    if (j == 0) {
                        f.bcCode = cfg.bcCodes[Face::jminus];
                        if (f.bcCode == BCCode::wall_with_slip || f.bcCode == BCCode::wall_with_slip) {
                            // Do not use ghost cell.
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j,k), -1};
                            f.cloud_nc = 1;
                        } else {
                            f.cells_in_cloud = {cfg.ghostCellIndex(Face::jminus,i,k,0), cfg.activeCellIndex(i,j,k)};
                            f.cloud_nc = 2;
                        }
                        f.faces_in_cloud = {cfg.iFaceIndex(i,j,k), cfg.iFaceIndex(i+1,j,k),
                                            cfg.kFaceIndex(i,j,k), cfg.kFaceIndex(i,j,k+1),
                                            -1, -1, -1, -1};
                        f.cloud_nf = 4;
                    } else if (j == cfg.njc) {
                        f.bcCode = cfg.bcCodes[Face::jplus];
                        if (f.bcCode == BCCode::wall_with_slip || f.bcCode == BCCode::wall_with_slip) {
                            // Do not use ghost cell.
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j-1,k), -1};
                            f.cloud_nc = 1;
                        } else {
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j-1,k), cfg.ghostCellIndex(Face::jplus,i,k,0)};
                            f.cloud_nc = 2;
                        }
                        f.faces_in_cloud = {cfg.iFaceIndex(i,j-1,k), cfg.iFaceIndex(i+1,j-1,k),
                                            cfg.kFaceIndex(i,j-1,k), cfg.kFaceIndex(i,j-1,k+1),
                                            -1, -1, -1, -1};
                        f.cloud_nf = 4;
                    } else {
                        f.bcCode = -1; // Interior face.
                        f.cells_in_cloud = {cfg.activeCellIndex(i,j-1,k), cfg.activeCellIndex(i,j,k)};
                        f.cloud_nc = 2;
                        f.faces_in_cloud = {cfg.iFaceIndex(i,j-1,k), cfg.iFaceIndex(i+1,j-1,k),
                                            cfg.kFaceIndex(i,j-1,k), cfg.kFaceIndex(i,j-1,k+1),
                                            cfg.iFaceIndex(i,j,k), cfg.iFaceIndex(i+1,j,k),
                                            cfg.kFaceIndex(i,j,k), cfg.kFaceIndex(i,j,k+1)};
                        f.cloud_nf = 8;
                    }
                }
            }
        }
        // kFaces
        for (int j=0; j < cfg.njc; j++) {
            for (int i=0; i < cfg.nic; i++) {
                for (int k=0; k < cfg.nkc+1; k++) {
                    FVFace& f = faces[cfg.kFaceIndex(i,j,k)];
                    f.vtx[0] = cfg.vtxIndex(i,j,k);
                    f.vtx[1] = cfg.vtxIndex(i+1,j,k);
                    f.vtx[2] = cfg.vtxIndex(i+1,j+1,k);
                    f.vtx[3] = cfg.vtxIndex(i,j+1,k);
                    // Set neighbouring cells for convective fluxes.
                    if (k == 0) {
                        f.left_cells[1] = cfg.ghostCellIndex(Face::kminus,i,j,1);
                        f.left_cells[0] = cfg.ghostCellIndex(Face::kminus,i,j,0);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i,j,k+1);
                        f.bcId = Face::kminus;
                        f.bcCode = cfg.bcCodes[Face::kminus];
                        if (f.bcCode == BCCode::inflow) f.inflowId = cfg.bc_fs[Face::kminus];
                        if (f.bcCode == BCCode::wall_no_slip_fixed_T) f.TWall = cfg.bc_TWall[Face::kminus];
                    } else if (k == 1) {
                        f.left_cells[1] = cfg.ghostCellIndex(Face::kminus,i,j,0);
                        f.left_cells[0] = cfg.activeCellIndex(i,j,k-1);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i,j,k+1);
                    } else if (k == cfg.nkc-1) {
                        f.left_cells[1] = cfg.activeCellIndex(i,j,k-2);
                        f.left_cells[0] = cfg.activeCellIndex(i,j,k-1);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.ghostCellIndex(Face::kplus,i,j,0);
                    } else if (k == cfg.nkc) {
                        f.left_cells[1] = cfg.activeCellIndex(i,j,k-2);
                        f.left_cells[0] = cfg.activeCellIndex(i,j,k-1);
                        f.right_cells[0] = cfg.ghostCellIndex(Face::kplus,i,j,0);
                        f.right_cells[1] = cfg.ghostCellIndex(Face::kplus,i,j,1);
                        f.bcId = Face::kplus;
                        f.bcCode = cfg.bcCodes[Face::kplus];
                        if (f.bcCode == BCCode::inflow) f.inflowId = cfg.bc_fs[Face::kplus];
                        if (f.bcCode == BCCode::wall_no_slip_fixed_T) f.TWall = cfg.bc_TWall[Face::kplus];
                    } else {
                        // All interior cells.
                        f.left_cells[1] = cfg.activeCellIndex(i,j,k-2);
                        f.left_cells[0] = cfg.activeCellIndex(i,j,k-1);
                        f.right_cells[0] = cfg.activeCellIndex(i,j,k);
                        f.right_cells[1] = cfg.activeCellIndex(i,j,k+1);
                    }
                    // Set cloud of FlowStates for gradient calculations of viscous fluxes.
                    if (k == 0) {
                        f.bcCode = cfg.bcCodes[Face::kminus];
                        if (f.bcCode == BCCode::wall_with_slip || f.bcCode == BCCode::wall_with_slip) {
                            // Do not use ghost cell.
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j,k), -1};
                            f.cloud_nc = 1;
                        } else {
                            f.cells_in_cloud = {cfg.ghostCellIndex(Face::kminus,i,j,0), cfg.activeCellIndex(i,j,k)};
                            f.cloud_nc = 2;
                        }
                        f.faces_in_cloud = {cfg.iFaceIndex(i,j,k), cfg.iFaceIndex(i+1,j,k),
                                            cfg.jFaceIndex(i,j,k), cfg.jFaceIndex(i,j+1,k),
                                            -1, -1, -1, -1};
                        f.cloud_nf = 4;
                    } else if (k == cfg.nkc) {
                        f.bcCode = cfg.bcCodes[Face::kplus];
                        if (f.bcCode == BCCode::wall_with_slip || f.bcCode == BCCode::wall_with_slip) {
                            // Do not use ghost cell.
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j,k-1), -1};
                            f.cloud_nc = 1;
                        } else {
                            f.cells_in_cloud = {cfg.activeCellIndex(i,j,k-1), cfg.ghostCellIndex(Face::kplus,i,j,0)};
                            f.cloud_nc = 2;
                        }
                        f.faces_in_cloud = {cfg.iFaceIndex(i,j,k-1), cfg.iFaceIndex(i+1,j,k-1),
                                            cfg.jFaceIndex(i,j,k-1), cfg.jFaceIndex(i,j+1,k-1),
                                            -1, -1, -1, -1};
                        f.cloud_nf = 4;
                    } else {
                        f.bcCode = -1; // Interior face.
                        f.cells_in_cloud = {cfg.activeCellIndex(i,j,k-1), cfg.activeCellIndex(i,j,k)};
                        f.cloud_nc = 2;
                        f.faces_in_cloud = {cfg.iFaceIndex(i,j,k-1), cfg.iFaceIndex(i+1,j,k-1),
                                            cfg.jFaceIndex(i,j,k-1), cfg.jFaceIndex(i,j+1,k-1),
                                            cfg.iFaceIndex(i,j,k), cfg.iFaceIndex(i+1,j,k),
                                            cfg.jFaceIndex(i,j,k), cfg.jFaceIndex(i,j+1,k)};
                        f.cloud_nf = 8;
                    }
                }
            }
        }
        return bytes_allocated;
    } // end configure()

    __host__
    void releaseMemory()
    {
        cells.resize(0);
        Q.resize(0);
        dQdt.resize(0);
        faces.resize(0);
        vertices.resize(0);
#ifdef CUDA
        if (cells_on_gpu) { hipFree(&cells_on_gpu); cells_on_gpu = NULL; }
        if (Q_on_gpu) { hipFree(&Q_on_gpu); Q_on_gpu = NULL; }
        if (dQdt_on_gpu) { hipFree(&dQdt_on_gpu); dQdt_on_gpu = NULL; }
        if (faces_on_gpu) { hipFree(&faces_on_gpu); faces_on_gpu = NULL; }
        if (vertices_on_gpu) { hipFree(&vertices_on_gpu); vertices_on_gpu = NULL; }
#endif
        return;
    }

    __host__
    void computeGeometry(const BConfig& cfg)
    // Compute cell and face geometric data.
    // Do this after reading the grid and flow files because we need the vertex locations
    // and because cell positions and volumes are part of the flow data.
    // This function will overwrite them with (potentially) better values.
    {
        for (int ic=0; ic < cfg.nActiveCells; ic++) {
            FVCell& c = cells[ic];
            hex_cell_properties(vertices[c.vtx[0]], vertices[c.vtx[1]],
                                vertices[c.vtx[2]], vertices[c.vtx[3]],
                                vertices[c.vtx[4]], vertices[c.vtx[5]],
                                vertices[c.vtx[6]], vertices[c.vtx[7]],
                                false, c.pos, c.volume, c.iLength, c.jLength, c.kLength);
        }
        for (auto& f : faces) {
            quad_properties(vertices[f.vtx[0]], vertices[f.vtx[1]],
                            vertices[f.vtx[2]], vertices[f.vtx[3]],
                            f.pos, f.n, f.t1, f.t2, f.area);
        }
        //
        if (!cfg.active) return; // No ghost cells for an inactive block.
        //
        // Work around the boundaries and extrapolate cell positions and lengths
        // into the ghost cells.  We need this data for high-order reconstruction
        // for the inviscid fluxes and for computation of the flow-property gradients
        // for the viscous fluxes.
        //
        // Face::iminus
        for (int k=0; k < cfg.nkc; k++) {
            for (int j=0; j < cfg.njc; j++) {
                FVFace& f = faces[cfg.iFaceIndex(0,j,k)];
                FVCell& c0 = cells[f.right_cells[0]];
                FVCell& g0 = cells[f.left_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.left_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::iplus
        for (int k=0; k < cfg.nkc; k++) {
            for (int j=0; j < cfg.njc; j++) {
                FVFace& f = faces[cfg.iFaceIndex(cfg.nic,j,k)];
                FVCell& c0 = cells[f.left_cells[0]];
                FVCell& g0 = cells[f.right_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.right_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::jminus
        for (int k=0; k < cfg.nkc; k++) {
            for (int i=0; i < cfg.nic; i++) {
                FVFace& f = faces[cfg.jFaceIndex(i,0,k)];
                FVCell& c0 = cells[f.right_cells[0]];
                FVCell& g0 = cells[f.left_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.left_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::jplus
        for (int k=0; k < cfg.nkc; k++) {
            for (int i=0; i < cfg.nic; i++) {
                FVFace& f = faces[cfg.jFaceIndex(i,cfg.njc,k)];
                FVCell& c0 = cells[f.left_cells[0]];
                FVCell& g0 = cells[f.right_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.right_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::kminus
        for (int j=0; j < cfg.njc; j++) {
            for (int i=0; i < cfg.nic; i++) {
                FVFace& f = faces[cfg.kFaceIndex(i,j,0)];
                FVCell& c0 = cells[f.right_cells[0]];
                FVCell& g0 = cells[f.left_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.left_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::kplus
        for (int j=0; j < cfg.njc; j++) {
            for (int i=0; i < cfg.nic; i++) {
                FVFace& f = faces[cfg.kFaceIndex(i,j,cfg.nkc)];
                FVCell& c0 = cells[f.left_cells[0]];
                FVCell& g0 = cells[f.right_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.right_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        //
        return;
    } // end computeGeometry()

    __host__
    void readGrid(const BConfig& cfg, string fileName, bool binary_data, bool vtkHeader=false)
    // Reads the vertex locations from a compressed file, resizing storage as needed.
    // The numbers of cells are also checked.
    {
        auto f = bxz::ifstream(fileName); // gzip file
        if (!f) {
            throw runtime_error("Did not open grid file successfully: "+fileName);
        }
        constexpr int maxc = 256;
        char line[maxc];
        int niv, njv, nkv;
        if (vtkHeader) {
            f.getline(line, maxc); // expect "vtk"
            f.getline(line, maxc); // title line
            f.getline(line, maxc); // expect "ASCII"
            f.getline(line, maxc); // expect "STRUCTURED_GRID"
            f.getline(line, maxc); // DIMENSIONS line
            sscanf(line, "DIMENSIONS %d %d %d", &niv, &njv, &nkv);
        } else {
            f.getline(line, maxc); // expect "structured_grid 1.0"
            f.getline(line, maxc); // label:
            f.getline(line, maxc); // dimensions:
            f.getline(line, maxc);
            sscanf(line, "niv: %d", &niv);
            f.getline(line, maxc);
            sscanf(line, "njv: %d", &njv);
            f.getline(line, maxc);
            sscanf(line, "nkv: %d", &nkv);
        }
        if ((cfg.nic != niv-1) || (cfg.njc != njv-1) || (cfg.nkc != nkv-1)) {
            throw runtime_error("Unexpected grid size: niv="+to_string(niv)+
                                " njv="+to_string(njv)+ " nkv="+to_string(nkv));
        }
        if (vertices.size() != niv*njv*nkv) throw runtime_error("Incorrect size of vertices.");
        //
        // Standard order of vertices.
        for (int k=0; k < nkv; k++) {
            for (int j=0; j < njv; j++) {
                for (int i=0; i < niv; i++) {
                    f.getline(line, maxc);
                    number x, y, z;
                    #ifdef FLOAT_NUMBERS
                    sscanf(line "%f %f %f", &x, &y, &z);
                    #else
                    sscanf(line, "%lf %lf %lf", &x, &y, &z);
                    #endif
                    vertices[cfg.vtxIndex(i,j,k)].set(x, y, z);
                } // for i
            } // for j
        } // for k
        f.close();
        return;
    } // end readGrid()

    __host__
    void readFlow(const BConfig& cfg, string fileName, bool binary_data)
    // Reads the flow data archive from a ZIP file.
    // The correct data storage is presumed to exist.
    //
    // Code modelled on the simple example by Dodrigo Rivas Costa found at
    // https://stackoverflow.com/questions/10440113/simple-way-to-unzip-a-zip-file-using-zlib
    {
        int err = 0;
        zip *z = zip_open(fileName.c_str(), ZIP_RDONLY, &err);
        if (err) {
            cerr << "Failed to open zip archive for reading: " << fileName << endl;
        }
        if (z) {
            struct zip_stat st;
            for (int m=0; m < IOvar::n; m++) {
                string name = IOvar::names[m];
                // Search archive for a variable's data.
                zip_stat_init(&st);
                zip_stat(z, name.c_str(), 0, &st);
                // Allocate enough memory for the uncompressed content and read it.
                char* content = new char[st.size];
                zip_file* f = zip_fopen(z, name.c_str(), 0);
                if (f) {
                    zip_fread(f, content, st.size);
                    zip_fclose(f);
                    stringstream ss(content);
                    string item;
                    for (int k=0; k < cfg.nkc; k++) {
                        for (int j=0; j < cfg.njc; j++) {
                            for (int i=0; i < cfg.nic; i++) {
                                getline(ss, item, '\n');
                                FVCell& c = cells[cfg.activeCellIndex(i,j,k)];
                                c.iovar_set(m, stod(item));
                            }
                        }
                    }
                } else {
                    cerr << "Could not open file " << name << " in ZIP archive " << fileName << endl;
                }
                delete[] content;
            }
            zip_close(z);
        }
        return;
    } // end readFlow()

    __host__
    void writeFlow(const BConfig& cfg, string fileName, bool binary_data)
    // Writes the flow data into a new ZIP archive file.
    // Any necessary directories are presumed to exist.
    {
        vector<string> data; // A place to retain the string data while the zip file is constructed.
        int err = 0;
        zip *z = zip_open(fileName.c_str(), ZIP_CREATE, &err);
        if (err) {
            cerr << "Failed to open zip archive for writing: " << fileName << endl;
        }
        if (z) {
            for (int m=0; m < IOvar::n; m++) {
                string name = IOvar::names[m];
                ostringstream ss;
                for (int k=0; k < cfg.nkc; k++) {
                    for (int j=0; j < cfg.njc; j++) {
                        for (int i=0; i < cfg.nic; i++) {
                            FVCell& c = cells[cfg.activeCellIndex(i,j,k)];
                            ss << c.iovar_get(m) << endl;
                        }
                    }
                }
                data.push_back(ss.str());
                int last = data.size()-1;
                // Add the data to the ZIP archive as a file.
                zip_source_t* zs = zip_source_buffer(z, data[last].c_str(), data[last].size(), 0);
                if (zs) {
                    int zindx = zip_file_add(z, name.c_str(), zs, ZIP_FL_OVERWRITE|ZIP_FL_ENC_UTF_8);
                    if (zindx < 0) {
                        cerr << "Could not add file " << name << " to ZIP archive " << fileName << endl;
                        zip_source_free(zs);
                    }
                } else {
                    cerr << "Error getting source to add file to zip: " << string(zip_strerror(z)) << endl;
                }
            }
            zip_close(z);
        }
        data.resize(0);
        return;
    } // end writeFlow()

    __host__
    number estimate_allowed_dt(const BConfig& cfg, number cfl)
    {
        number smallest_dt = numeric_limits<number>::max();
        for (int i=0; i < cfg.nActiveCells; i++) {
            FVCell& c = cells[i];
            Vector3 inorm = faces[c.face[Face::iminus]].n;
            Vector3 jnorm = faces[c.face[Face::jminus]].n;
            Vector3 knorm = faces[c.face[Face::kminus]].n;
            smallest_dt = fmin(smallest_dt, c.estimate_local_dt(inorm, jnorm, knorm, cfl));
        }
        return smallest_dt;
    } // end estimate_allowed_dt()

    __host__
    void encodeConserved(const BConfig& cfg, int level)
    {
        for (int i=0; i < cfg.nActiveCells; i++) {
            FlowState& fs = cells[i].fs;
            ConservedQuantities& U = Q[level*cfg.nActiveCells + i];
            fs.encode_conserved(U);
        }
    }

    __host__
    int decodeConserved(const BConfig& cfg, int level)
    {
        int bad_cell_count = 0;
        for (int i=0; i < cfg.nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities U = Q[level*cfg.nActiveCells + i];
            int bad_cell_flag = c.fs.decode_conserved(U);
            bad_cell_count += bad_cell_flag;
            if (bad_cell_flag) {
                cerr << "DEBUG-A Bad cell at pos=" << c.pos.toString() << endl;
            }
        }
        return bad_cell_count;
    }

    __host__
    void calculate_convective_fluxes(int flux_calc, int x_order)
    {
        for (auto& face : faces) {
            FlowState& fsL1 = cells[face.left_cells[1]].fs;
            FlowState& fsL0 = cells[face.left_cells[0]].fs;
            FlowState& fsR0 = cells[face.right_cells[0]].fs;
            FlowState& fsR1 = cells[face.right_cells[1]].fs;
            face.calculate_convective_flux(fsL1, fsL0, fsR0, fsR1, flux_calc, x_order);
        }
    } // end calculate_convective_fluxes()

    __host__
    void setup_LSQ_arrays()
    {
        int failures = 0;
        for (auto& face : faces) {
            int flag = setup_LSQ_arrays_at_face(face, cells.data(), faces.data());
            if (flag) {
                cerr << "Singular normal matrix at f.pos=" << face.pos.toString() << endl;
            }
        }
        if (failures > 0) {
            throw runtime_error("Singular matrices encountered while setting up LSQ weights.");
        }
    }

    __host__
    void add_viscous_flux()
    {
        for (auto& face : faces) {
            calculate_gradients_at_face(face, cells.data(), faces.data());
            face.add_viscous_flux();
        }
    }

    __host__
    int update_stage_1(const BConfig& cfg, number dt)
    // Stage 1 of the TVD-RK3 update scheme (predictor step).
    {
        int bad_cell_count = 0;
        for (int i=0; i < cfg.nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities& dUdt0 = dQdt[i];
            c.eval_dUdt(dUdt0, faces.data());
            ConservedQuantities& U0 = Q[i];
            ConservedQuantities& U1 = Q[cfg.nActiveCells + i];
            for (int j=0; j < CQI::n; j++) {
                U1[j] = U0[j] + dt*dUdt0[j];
            }
            int bad_cell_flag = c.fs.decode_conserved(U1);
            bad_cell_count += bad_cell_flag;
            if (bad_cell_flag) {
                cerr << "Stage 1 update, Bad cell at pos=" << c.pos.toString() << endl;
            }
        }
        return bad_cell_count;
    } // end update_stage_1()

    __host__
    int update_stage_2(const BConfig& cfg, number dt)
    // Stage 2 of the TVD-RK3 update scheme.
    {
        int bad_cell_count = 0;
        for (int i=0; i < cfg.nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities& dUdt0 = dQdt[i];
            ConservedQuantities& dUdt1 = dQdt[cfg.nActiveCells + i];
            c.eval_dUdt(dUdt1, faces.data());
            ConservedQuantities& U0 = Q[i];
            ConservedQuantities& U1 = Q[cfg.nActiveCells + i];
            for (int j=0; j < CQI::n; j++) {
                U1[j] = U0[j] + 0.25*dt*(dUdt0[j] + dUdt1[j]);
            }
            int bad_cell_flag = c.fs.decode_conserved(U1);
            bad_cell_count += bad_cell_flag;
            if (bad_cell_flag) {
                cerr << "Stage 2 update, Bad cell at pos=" << c.pos.toString() << endl;
            }
        }
        return bad_cell_count;
    } // end update_stage_2()

    __host__
    int update_stage_3(const BConfig& cfg, number dt)
    // Stage 3 of the TVD_RK3 update scheme.
    {
        int bad_cell_count = 0;
        for (int i=0; i < cfg.nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities& dUdt0 = dQdt[i];
            ConservedQuantities& dUdt1 = dQdt[cfg.nActiveCells + i];
            ConservedQuantities& dUdt2 = dQdt[2*cfg.nActiveCells + i];
            c.eval_dUdt(dUdt2, faces.data());
            ConservedQuantities& U0 = Q[i];
            ConservedQuantities& U1 = Q[cfg.nActiveCells + i];
            for (int j=0; j < CQI::n; j++) {
                U1[j] = U0[j] + dt*(1.0/6.0*dUdt0[j] + 1.0/6.0*dUdt1[j] + 4.0/6.0*dUdt2[j]);
            }
            int bad_cell_flag = c.fs.decode_conserved(U1);
            bad_cell_count += bad_cell_flag;
            if (bad_cell_flag) {
                cerr << "Stage 3 update, Bad cell at pos=" << c.pos.toString() << endl;
            }
        }
        return bad_cell_count;
    } // end update_stage_3()

    __host__
    void copy_conserved_data(const BConfig& cfg, int from_level, int to_level)
    {
        for (auto i=0; i < cfg.nActiveCells; i++) {
            ConservedQuantities& U_from = Q[from_level*cfg.nActiveCells + i];
            ConservedQuantities& U_to = Q[to_level*cfg.nActiveCells + i];
            for (int j=0; j < CQI::n; j++) {
                U_to[j] = U_from[j];
            }
        }
    } // end copy_conserved_data()

}; // end Block



// GPU global functions cannot be member functions of FluidBlock
// so we need to pass the FluidBlock reference into them and that
// Block struct also needs to be in the global memory of the GPU.

__global__
void estimate_allowed_dt_on_gpu(Block& blk, const BConfig& cfg, number cfl, long long int* smallest_dt_picos)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nActiveCells) {
        FVCell& c = blk.cells_on_gpu[i];
        Vector3 inorm = blk.faces_on_gpu[c.face[Face::iminus]].n;
        Vector3 jnorm = blk.faces_on_gpu[c.face[Face::jminus]].n;
        Vector3 knorm = blk.faces_on_gpu[c.face[Face::kminus]].n;
        long long int dt_picos = trunc(c.estimate_local_dt(inorm, jnorm, knorm, cfl)*1.0e12);
        atomicMin(smallest_dt_picos, dt_picos);
    }
}

__global__
void encodeConserved_on_gpu(Block& blk, const BConfig& cfg, int level)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nActiveCells) {
        FlowState& fs = blk.cells_on_gpu[i].fs;
        ConservedQuantities& U = blk.Q_on_gpu[level*cfg.nActiveCells + i];
        fs.encode_conserved(U);
    }
}

__global__
void copy_conserved_data_on_gpu(Block& blk, const BConfig& cfg, int from_level, int to_level)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nActiveCells) {
        ConservedQuantities& U_from = blk.Q_on_gpu[from_level*cfg.nActiveCells + i];
        ConservedQuantities& U_to = blk.Q_on_gpu[to_level*cfg.nActiveCells + i];
        for (int j=0; j < CQI::n; j++) {
            U_to[j] = U_from[j];
        }
    }
}

__global__
void calculate_convective_fluxes_on_gpu(Block& blk, const BConfig& cfg, int flux_calc, int x_order)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nFaces) {
        FVFace& face = blk.faces_on_gpu[i];
        FlowState& fsL1 = blk.cells_on_gpu[face.left_cells[1]].fs;
        FlowState& fsL0 = blk.cells_on_gpu[face.left_cells[0]].fs;
        FlowState& fsR0 = blk.cells_on_gpu[face.right_cells[0]].fs;
        FlowState& fsR1 = blk.cells_on_gpu[face.right_cells[1]].fs;
        face.calculate_convective_flux(fsL1, fsL0, fsR0, fsR1, flux_calc, x_order);
    }
}

__global__
void setup_LSQ_arrays_on_gpu(Block& blk, const BConfig& cfg, int* failures)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nFaces) {
        FVFace& face = blk.faces_on_gpu[i];
        int flag = setup_LSQ_arrays_at_face(face, blk.cells_on_gpu, blk.faces_on_gpu);
        atomicAdd(failures, flag);
    }
}

__global__
void add_viscous_flux_on_gpu(Block& blk, const BConfig& cfg)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nFaces) {
        FVFace& face = blk.faces_on_gpu[i];
        calculate_gradients_at_face(face, blk.cells_on_gpu, blk.faces_on_gpu);
        face.add_viscous_flux();
    }
}

__global__
void update_stage_1_on_gpu(Block& blk, const BConfig& cfg, number dt, int* bad_cell_count)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nActiveCells) {
        FVCell& c = blk.cells_on_gpu[i];
        ConservedQuantities& dUdt0 = blk.dQdt_on_gpu[i];
        c.eval_dUdt(dUdt0, blk.faces_on_gpu);
        ConservedQuantities& U0 = blk.Q_on_gpu[i];
        ConservedQuantities& U1 = blk.Q_on_gpu[cfg.nActiveCells + i];
        for (int j=0; j < CQI::n; j++) {
            U1[j] = U0[j] + dt*dUdt0[j];
        }
        int bad_cell_flag = c.fs.decode_conserved(U1);
        atomicAdd(bad_cell_count, bad_cell_flag);
        if (bad_cell_flag) {
            printf("Stage 1 update, Bad cell at pos x=%g y=%g z=%g\n", c.pos.x, c.pos.y, c.pos.z);
        }
    }
}

__global__
void update_stage_2_on_gpu(Block& blk, const BConfig& cfg, number dt, int* bad_cell_count)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nActiveCells) {
        FVCell& c = blk.cells_on_gpu[i];
        ConservedQuantities& dUdt0 = blk.dQdt_on_gpu[i];
        ConservedQuantities& dUdt1 = blk.dQdt_on_gpu[cfg.nActiveCells + i];
        c.eval_dUdt(dUdt1, blk.faces_on_gpu);
        ConservedQuantities& U0 = blk.Q_on_gpu[i];
        ConservedQuantities& U1 = blk.Q_on_gpu[cfg.nActiveCells + i];
        for (int j=0; j < CQI::n; j++) {
            U1[j] = U0[j] + 0.25*dt*(dUdt0[j] + dUdt1[j]);
        }
        int bad_cell_flag = c.fs.decode_conserved(U1);
        atomicAdd(bad_cell_count, bad_cell_flag);
        if (bad_cell_flag) {
            printf("Stage 2 update, Bad cell at pos x=%g y=%g z=%g\n", c.pos.x, c.pos.y, c.pos.z);
        }
    }
}

__global__
void update_stage_3_on_gpu(Block& blk, const BConfig& cfg, number dt, int* bad_cell_count)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < cfg.nActiveCells) {
        FVCell& c = blk.cells_on_gpu[i];
        ConservedQuantities& dUdt0 = blk.dQdt_on_gpu[i];
        ConservedQuantities& dUdt1 = blk.dQdt_on_gpu[cfg.nActiveCells + i];
        ConservedQuantities& dUdt2 = blk.dQdt_on_gpu[2*cfg.nActiveCells + i];
        c.eval_dUdt(dUdt2, blk.faces_on_gpu);
        ConservedQuantities& U0 = blk.Q_on_gpu[i];
        ConservedQuantities& U1 = blk.Q_on_gpu[cfg.nActiveCells + i];
        for (int j=0; j < CQI::n; j++) {
            U1[j] = U0[j] + dt*(1.0/6.0*dUdt0[j] + 1.0/6.0*dUdt1[j] + 4.0/6.0*dUdt2[j]);
        }
        int bad_cell_flag = c.fs.decode_conserved(U1);
        atomicAdd(bad_cell_count, bad_cell_flag);
        if (bad_cell_flag) {
            printf("Stage 3 update, Bad cell at pos x=%g y=%g z=%g\n", c.pos.x, c.pos.y, c.pos.z);
        }
    }
}

#endif
