#include "hip/hip_runtime.h"
// block.cu
// Include file for chicken.
// PJ 2022-09-11

#ifndef BLOCK_INCLUDED
#define BLOCK_INCLUDED

#include <string>
#include <fstream>
#include <sstream>
#include <stdexcept>
#include <limits>
#include "include/bxzstr/bxzstr.hpp"
#include <zip.h>

#include "number.cu"
#include "vector3.cu"
#include "config.cu"
#include "gas.cu"
#include "vertex.cu"
#include "flow.cu"
#include "face.cu"
#include "cell.cu"
#include "flux.cu"

using namespace std;

struct Block {
    // Active cells are the "real" cells in the simulation.
    // We compute the evolution of the gas-dynamic flow properties within them.
    int nic; // Number of cells i-direction.
    int njc; // Number of cells j-direction.
    int nkc; // Number of cells k-direction.
    int nActiveCells; // Number of active cells (with conserved quantities) in the block.
    vector<FVCell> cells;
    //
    // Active cells have conserved quantities data, along with the time derivatives.
    vector<ConservedQuantities> Q;
    vector<ConservedQuantities> dQdt;
    //
    // Ghost cells are associated with each block boundary face and
    // will be stored at the end of the active cells collection.
    // The flux calculation functions will dip into this collection for
    // active cells and ghost cells without knowing the difference.
    // Also, for each boundary face, we store some config information
    // to make the boundary-condition code a bit more compact.
    array<int,6> n0c; // number of cells in first index direction for each face.
    array<int,6> n1c; // Number of cells in second index direction for each face.
    array<int,6> nGhostCells; // Number of ghost cells on each face.
    array<int,6> firstGhostCells; // Index of the first ghost cell for each face.
    //
    // Collections of faces which bound the active cells.
    // We compute fluxes of conserved flow properties across these faces.
    vector<FVFace> iFaces;
    vector<FVFace> jFaces;
    vector<FVFace> kFaces;
    //
    // The vertices are used to define the locations and geometric properties
    // of faces and cells.
    vector<Vector3> vertices;

    __host__
    string toString() {
        string repr = "Block(nic=" + to_string(nic) +
            ", njc=" + to_string(njc) + ", nkc=" + to_string(nkc) + ")";
        return repr;
    }

    // Methods to index the elements making up the block.

    __host__ __device__
    int activeCellIndex(int i, int j, int k)
    {
        return k*nic*njc + j*nic + i;
    }

    __host__ __device__
    int ghostCellIndex(int faceIndx, int i0, int i1, int depth)
    {
        int cellIndxOnFace = i1*n0c[faceIndx] + i0;
        int nCellsOnFace = n0c[faceIndx]*n1c[faceIndx];
        return firstGhostCells[faceIndx] + nCellsOnFace*depth + cellIndxOnFace;
    }

    __host__ __device__
    int iFaceIndex(int i, int j, int k)
    {
        return i*njc*nkc + k*njc + j;
    }

    __host__ __device__
    int jFaceIndex(int i, int j, int k)
    {
        return j*nic*nkc + k*nic + i;
    }

    __host__ __device__
    int kFaceIndex(int i, int j, int k)
    {
        return k*nic*njc + j*nic + i;
    }

    __host__ __device__
    int vtxIndex(int i, int j, int k)
    {
        return k*(nic+1)*(njc+1) + j*(nic+1) + i;
    }

    __host__
    void configure(int i, int j, int k)
    // Set up the block to hold the grid and flow data.
    // Do this before reading a grid or flow file.
    {
        nic = i;
        njc = j;
        nkc = k;
        int nActiveCells = nic*njc*nkc;
        //
        // For the moment assume that all boundary conditions require ghost cells.
        n0c[Face::iminus] = njc; n1c[Face::iminus] = nkc;
        n0c[Face::iplus] = njc; n1c[Face::iplus] = nkc;
        n0c[Face::jminus] = nic; n1c[Face::jminus] = nkc;
        n0c[Face::jplus] = nic; n1c[Face::jplus] = nkc;
        n0c[Face::kminus] = nic; n1c[Face::kminus] = njc;
        n0c[Face::kplus] = nic; n1c[Face::kplus] = njc;
        for (int ib=0; ib < 6; ib++) {
            nGhostCells[ib] = 2*n0c[ib]*n1c[ib];
            if (ib > 1) {
                firstGhostCells[ib] = firstGhostCells[ib-1] + nGhostCells[ib-1];
            } else {
                firstGhostCells[ib] = nActiveCells;
            }
        }
        //
        // Now that we know the numbers of cells, resize the data store to fit them all.
        cells.resize(firstGhostCells[5]+nGhostCells[5]);
        Q.resize(nActiveCells*TLevels);
        dQdt.resize(nActiveCells*TLevels);
        #ifdef CUDA
        // We need to allocate corresponding memory space on the GPU.
        // [TODO]
        #endif
        //
        // Each set of finite-volume faces is in the index-plane of the corresponding vertices.
        iFaces.resize((nic+1)*njc*nkc);
        jFaces.resize(nic*(njc+1)*nkc);
        kFaces.resize(nic*njc*(nkc+1));
        //
        // And the vertices.
        vertices.resize((nic+1)*(njc+1)*(nkc+1));
        //
        // Make connections from cells to faces and vertices.
        for (int k=0; k < nkc; k++) {
            for (int j=0; j < njc; j++) {
                for (int i=0; i < nic; i++) {
                    FVCell& c = cells[activeCellIndex(i,j,k)];
                    c.face[Face::iminus] = iFaceIndex(i,j,k);
                    c.face[Face::iplus] = iFaceIndex(i+1,j,k);
                    c.face[Face::jminus] = jFaceIndex(i,j,k);
                    c.face[Face::jplus] = jFaceIndex(i,j+1,k);
                    c.face[Face::kminus] = kFaceIndex(i,j,k);
                    c.face[Face::kplus] = kFaceIndex(i,j,k+1);
                    c.vtx[0] = vtxIndex(i,j,k);
                    c.vtx[1] = vtxIndex(i+1,j,k);
                    c.vtx[2] = vtxIndex(i+1,j+1,k);
                    c.vtx[3] = vtxIndex(i,j+1,k);
                    c.vtx[4] = vtxIndex(i,j,k+1);
                    c.vtx[5] = vtxIndex(i+1,j,k+1);
                    c.vtx[6] = vtxIndex(i+1,j+1,k+1);
                    c.vtx[7] = vtxIndex(i,j+1,k+1);
                }
            }
        }
        //
        // Make connections from faces to cells and vertices.
        // iFaces
        for (int k=0; k < nkc; k++) {
            for (int j=0; j < njc; j++) {
                for (int i=0; i < nic+1; i++) {
                    FVFace& f = iFaces[iFaceIndex(i,j,k)];
                    f.vtx[0] = vtxIndex(i,j,k);
                    f.vtx[1] = vtxIndex(i,j+1,k);
                    f.vtx[2] = vtxIndex(i,j+1,k+1);
                    f.vtx[3] = vtxIndex(i,j,k+1);
                    if (i == 0) {
                        f.left_cells[0] = ghostCellIndex(Face::iminus,j,k,1);
                        f.left_cells[1] = ghostCellIndex(Face::iminus,j,k,0);
                        f.right_cells[0] = activeCellIndex(i+1,j,k);
                        f.right_cells[1] = activeCellIndex(i+2,j,k);
                    } else if (i == 1) {
                        f.left_cells[0] = ghostCellIndex(Face::iminus,j,k,0);
                        f.left_cells[1] = activeCellIndex(i-1,j,k);
                        f.right_cells[0] = activeCellIndex(i+1,j,k);
                        f.right_cells[1] = activeCellIndex(i+2,j,k);
                    } else if (i == nic-1) {
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i-1,j,k);
                        f.right_cells[0] = activeCellIndex(i+1,j,k);
                        f.right_cells[1] = ghostCellIndex(Face::iplus,j,k,0);
                    } else if (i == nic) {
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i-1,j,k);
                        f.right_cells[0] = ghostCellIndex(Face::iplus,j,k,0);
                        f.right_cells[1] = ghostCellIndex(Face::iplus,j,k,1);
                    } else {
                        // Interior cell.
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i-1,j,k);
                        f.right_cells[0] = activeCellIndex(i+1,j,k);
                        f.right_cells[1] = activeCellIndex(i+2,j,k);
                    }
                }
            }
        }
        // jFaces
        for (int k=0; k < nkc; k++) {
            for (int i=0; i < nic; i++) {
                for (int j=0; j < njc+1; j++) {
                    FVFace& f = jFaces[jFaceIndex(i,j,k)];
                    f.vtx[0] = vtxIndex(i,j,k);
                    f.vtx[1] = vtxIndex(i+1,j,k);
                    f.vtx[2] = vtxIndex(i+1,j,k+1);
                    f.vtx[3] = vtxIndex(i,j,k+1);
                    if (j == 0) {
                        f.left_cells[0] = ghostCellIndex(Face::jminus,i,k,1);
                        f.left_cells[1] = ghostCellIndex(Face::jminus,i,k,0);
                        f.right_cells[0] = activeCellIndex(i,j+1,k);
                        f.right_cells[1] = activeCellIndex(i,j+2,k);
                    } else if (j == 1) {
                        f.left_cells[0] = ghostCellIndex(Face::jminus,i,k,0);
                        f.left_cells[1] = activeCellIndex(i,j-1,k);
                        f.right_cells[0] = activeCellIndex(i,j+1,k);
                        f.right_cells[1] = activeCellIndex(i,j+2,k);
                    } else if (j == njc-1) {
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i,j-1,k);
                        f.right_cells[0] = activeCellIndex(i,j+1,k);
                        f.right_cells[1] = ghostCellIndex(Face::jplus,i,k,0);
                    } else if (j == njc) {
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i,j-1,k);
                        f.right_cells[0] = ghostCellIndex(Face::jplus,i,k,0);
                        f.right_cells[1] = ghostCellIndex(Face::jplus,i,k,1);
                    } else {
                        // Interior cell.
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i,j-1,k);
                        f.right_cells[0] = activeCellIndex(i,j+1,k);
                        f.right_cells[1] = activeCellIndex(i,j+2,k);
                    }
                }
            }
        }
        // kFaces
        for (int j=0; j < njc; j++) {
            for (int i=0; i < nic; i++) {
                for (int k=0; k < nkc+1; k++) {
                    FVFace& f = kFaces[kFaceIndex(i,j,k)];
                    f.vtx[0] = vtxIndex(i,j,k);
                    f.vtx[1] = vtxIndex(i+1,j,k);
                    f.vtx[2] = vtxIndex(i+1,j+1,k);
                    f.vtx[3] = vtxIndex(i,j+1,k);
                    if (k == 0) {
                        f.left_cells[0] = ghostCellIndex(Face::kminus,i,j,1);
                        f.left_cells[1] = ghostCellIndex(Face::kminus,i,j,0);
                        f.right_cells[0] = activeCellIndex(i,j,k+1);
                        f.right_cells[1] = activeCellIndex(i,j,k+2);
                    } else if (k == 1) {
                        f.left_cells[0] = ghostCellIndex(Face::kminus,i,j,0);
                        f.left_cells[1] = activeCellIndex(i,j,k-1);
                        f.right_cells[0] = activeCellIndex(i,j,k+1);
                        f.right_cells[1] = activeCellIndex(i,j,k+2);
                    } else if (k == nkc-1) {
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i,j,k-1);
                        f.right_cells[0] = activeCellIndex(i,j,k+1);
                        f.right_cells[1] = ghostCellIndex(Face::kplus,i,j,0);
                    } else if (k == nkc) {
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i,j,k-1);
                        f.right_cells[0] = ghostCellIndex(Face::kplus,i,j,0);
                        f.right_cells[1] = ghostCellIndex(Face::kplus,i,j,1);
                    } else {
                        // Interior cell.
                        f.left_cells[0] = activeCellIndex(i,j,k);
                        f.left_cells[1] = activeCellIndex(i,j,k-1);
                        f.right_cells[0] = activeCellIndex(i,j,k+1);
                        f.right_cells[1] = activeCellIndex(i,j,k+2);
                    }
                }
            }
        }
        return;
    } // end configure()

    __host__
    void computeGeometry()
    // Compute cell and face geometric data.
    // Do this after reading the grid and flow files because we need the vertex locations
    // and because cell positions and volumes are part of the flow data.
    // This function will overwrite them with (potentially) better values.
    {
        for (int k=0; k < nkc; k++) {
            for (int j=0; j < njc; j++) {
                for (int i=0; i < nic; i++) {
                    FVCell& c = cells[activeCellIndex(i,j,k)];
                    hex_cell_properties(vertices[c.vtx[0]], vertices[c.vtx[1]],
                                        vertices[c.vtx[2]], vertices[c.vtx[3]],
                                        vertices[c.vtx[4]], vertices[c.vtx[5]],
                                        vertices[c.vtx[6]], vertices[c.vtx[7]],
                                        false, c.pos, c.volume, c.iLength, c.jLength, c.kLength);
                }
            }
        }
        // iFaces
        for (int k=0; k < nkc; k++) {
            for (int j=0; j < njc; j++) {
                for (int i=0; i < nic+1; i++) {
                    FVFace& f = iFaces[iFaceIndex(i,j,k)];
                    quad_properties(vertices[f.vtx[0]], vertices[f.vtx[1]],
                                    vertices[f.vtx[2]], vertices[f.vtx[3]],
                                    f.pos, f.n, f.t1, f.t2, f.area);
                }
            }
        }
        // jFaces
        for (int k=0; k < nkc; k++) {
            for (int i=0; i < nic; i++) {
                for (int j=0; j < njc+1; j++) {
                    FVFace& f = jFaces[jFaceIndex(i,j,k)];
                    quad_properties(vertices[f.vtx[0]], vertices[f.vtx[1]],
                                    vertices[f.vtx[2]], vertices[f.vtx[3]],
                                    f.pos, f.n, f.t1, f.t2, f.area);
                }
            }
        }
        // kFaces
        for (int j=0; j < njc; j++) {
            for (int i=0; i < nic; i++) {
                for (int k=0; k < nkc+1; k++) {
                    FVFace& f = kFaces[kFaceIndex(i,j,k)];
                    quad_properties(vertices[f.vtx[0]], vertices[f.vtx[1]],
                                    vertices[f.vtx[2]], vertices[f.vtx[3]],
                                    f.pos, f.n, f.t1, f.t2, f.area);
                }
            }
        }
        //
        // Work around the boundaries and extrapolate cell positions and lengths
        // into the ghost cells.  We need this data for high-order reconstruction
        // for the inviscid fluxes and for computation of the flow-property gradients
        // for the viscous fluxes.
        //
        // Face::iminus
        for (int k=0; k < nkc; k++) {
            for (int j=0; j < njc; j++) {
                FVFace& f = iFaces[iFaceIndex(0,j,k)];
                FVCell& c0 = cells[f.right_cells[0]];
                FVCell& g0 = cells[f.left_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.left_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::iplus
        for (int k=0; k < nkc; k++) {
            for (int j=0; j < njc; j++) {
                FVFace& f = iFaces[iFaceIndex(nic,j,k)];
                FVCell& c0 = cells[f.left_cells[0]];
                FVCell& g0 = cells[f.right_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.right_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::jminus
        for (int k=0; k < nkc; k++) {
            for (int i=0; i < nic; i++) {
                FVFace& f = jFaces[jFaceIndex(i,0,k)];
                FVCell& c0 = cells[f.right_cells[0]];
                FVCell& g0 = cells[f.left_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.left_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::jplus
        for (int k=0; k < nkc; k++) {
            for (int i=0; i < nic; i++) {
                FVFace& f = jFaces[jFaceIndex(i,njc,k)];
                FVCell& c0 = cells[f.left_cells[0]];
                FVCell& g0 = cells[f.right_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.right_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::kminus
        for (int j=0; j < njc; j++) {
            for (int i=0; i < nic; i++) {
                FVFace& f = kFaces[kFaceIndex(i,j,0)];
                FVCell& c0 = cells[f.right_cells[0]];
                FVCell& g0 = cells[f.left_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.left_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        // Face::kplus
        for (int j=0; j < njc; j++) {
            for (int i=0; i < nic; i++) {
                FVFace& f = kFaces[kFaceIndex(i,j,nkc)];
                FVCell& c0 = cells[f.left_cells[0]];
                FVCell& g0 = cells[f.right_cells[0]];
                g0.iLength = c0.iLength;
                g0.jLength = c0.jLength;
                g0.kLength = c0.kLength;
                Vector3 d = f.pos; d.sub(c0.pos);
                g0.pos = f.pos; g0.pos.add(d);
                //
                FVCell& g1 = cells[f.right_cells[1]];
                g1.iLength = c0.iLength;
                g1.jLength = c0.jLength;
                g1.kLength = c0.kLength;
                d.mul(3.0);
                g1.pos = f.pos; g1.pos.add(d);
            }
        }
        //
        return;
    } // end computeGeometry()

    __host__
    void readGrid(string fileName, bool vtkHeader=false)
    // Reads the vertex locations from a compressed file, resizing storage as needed.
    // The numbers of cells are also checked.
    {
        auto f = bxz::ifstream(fileName); // gzip file
        if (!f) {
            throw runtime_error("Did not open grid file successfully: "+fileName);
        }
        constexpr int maxc = 256;
        char line[maxc];
        int niv, njv, nkv;
        if (vtkHeader) {
            f.getline(line, maxc); // expect "vtk"
            f.getline(line, maxc); // title line
            f.getline(line, maxc); // expect "ASCII"
            f.getline(line, maxc); // expect "STRUCTURED_GRID"
            f.getline(line, maxc); // DIMENSIONS line
            sscanf(line, "DIMENSIONS %d %d %d", &niv, &njv, &nkv);
        } else {
            f.getline(line, maxc); // expect "structured_grid 1.0"
            f.getline(line, maxc); // label:
            f.getline(line, maxc); // dimensions:
            f.getline(line, maxc);
            sscanf(line, "niv: %d", &niv);
            f.getline(line, maxc);
            sscanf(line, "njv: %d", &njv);
            f.getline(line, maxc);
            sscanf(line, "nkv: %d", &nkv);
        }
        if ((nic != niv-1) || (njc != njv-1) || (nkc != nkv-1)) {
            throw runtime_error("Unexpected grid size: niv="+to_string(niv)+
                                " njv="+to_string(njv)+ " nkv="+to_string(nkv));
        }
        vertices.resize(niv*njv*nkv);
        //
        // Standard order of vertices.
        for (int k=0; k < nkv; k++) {
            for (int j=0; j < njv; j++) {
                for (int i=0; i < niv; i++) {
                    f.getline(line, maxc);
                    number x, y, z;
                    #ifdef FLOAT_NUMBERS
                    sscanf(line "%f %f %f", &x, &y, &z);
                    #else
                    sscanf(line, "%lf %lf %lf", &x, &y, &z);
                    #endif
                    vertices[vtxIndex(i,j,k)].set(x, y, z);
                } // for i
            } // for j
        } // for k
        f.close();
        return;
    } // end readGrid()

    __host__
    void readFlow(string fileName)
    // Reads the flow data archive from a ZIP file.
    // The correct data storage is presumed to exist.
    //
    // Code modelled on the simple example by Dodrigo Rivas Costa found at
    // https://stackoverflow.com/questions/10440113/simple-way-to-unzip-a-zip-file-using-zlib
    {
        int err = 0;
        zip *z = zip_open(fileName.c_str(), ZIP_RDONLY, &err);
        if (err) {
            cerr << "Failed to open zip archive for reading: " << fileName << endl;
        }
        if (z) {
            struct zip_stat st;
            for (int m=0; m < IOvar::n; m++) {
                string name = IOvar::names[m];
                // Search archive for a variable's data.
                zip_stat_init(&st);
                zip_stat(z, name.c_str(), 0, &st);
                // Allocate enough memory for the uncompressed content and read it.
                char* content = new char[st.size];
                zip_file* f = zip_fopen(z, name.c_str(), 0);
                if (f) {
                    zip_fread(f, content, st.size);
                    zip_fclose(f);
                    stringstream ss(content);
                    string item;
                    for (int k=0; k < nkc; k++) {
                        for (int j=0; j < njc; j++) {
                            for (int i=0; i < nic; i++) {
                                getline(ss, item, '\n');
                                FVCell& c = cells[activeCellIndex(i,j,k)];
                                c.iovar_set(m, stod(item));
                            }
                        }
                    }
                } else {
                    cerr << "Could not open file " << name << " in ZIP archive " << fileName << endl;
                }
                delete[] content;
            }
            zip_close(z);
        }
        return;
    } // end readFlow()

    __host__
    void writeFlow(string fileName)
    // Writes the flow data into a new ZIP archive file.
    // Any necessary directories are presumed to exist.
    {
        int err = 0;
        zip *z = zip_open(fileName.c_str(), ZIP_CREATE, &err);
        if (err) {
            cerr << "Failed to open zip archive for writing: " << fileName << endl;
        }
        if (z) {
            for (int m=0; m < IOvar::n; m++) {
                string name = IOvar::names[m];
                ostringstream ss;
                for (int k=0; k < nkc; k++) {
                    for (int j=0; j < njc; j++) {
                        for (int i=0; i < nic; i++) {
                            FVCell& c = cells[activeCellIndex(i,j,k)];
                            ss << c.iovar_get(m) << endl;
                        }
                    }
                }
                string data = ss.str();
                // Add the data to the ZIP archive as a file.
                zip_source_t* zs = zip_source_buffer(z, data.c_str(), data.size(), 0);
                if (zs) {
                    int zindx = zip_file_add(z, name.c_str(), zs, ZIP_FL_OVERWRITE);
                    if (zindx < 0) {
                        cerr << "Could not add file " << name << " to ZIP archive " << fileName << endl;
                        zip_source_free(zs);
                    }
                } else {
                    cerr << "Error adding file to zip: " << string(zip_strerror(z)) << endl;
                    zip_source_free(zs);
                }
            }
            zip_close(z);
        }
        return;
    } // end writeFlow()

    __host__ __device__
    number estimate_local_dt(FVCell& c, Vector3 inorm, Vector3 jnorm, Vector3 knorm, number cfl)
    {
        // We assume that the cells are (roughly) hexagonal and work with
        // velocities normal to the faces.
        FlowState& fs = c.fs;
        number isignal = c.iLength/(fabs(fs.vel.dot(inorm))+fs.gas.a);
        number jsignal = c.jLength/(fabs(fs.vel.dot(jnorm))+fs.gas.a);
        number ksignal = c.kLength/(fabs(fs.vel.dot(knorm))+fs.gas.a);
        return cfl * fmin(fmin(isignal,jsignal),ksignal);
    } // end estimate_local_dt()

    __host__
    number estimate_allowed_dt(number cfl)
    {
        number smallest_dt = numeric_limits<number>::max();
        for (auto i=0; i < nActiveCells; i++) {
            FVCell& c = cells[i];
            Vector3 inorm = iFaces[c.face[Face::iminus]].n;
            Vector3 jnorm = jFaces[c.face[Face::jminus]].n;
            Vector3 knorm = kFaces[c.face[Face::kminus]].n;
            smallest_dt = fmin(smallest_dt, estimate_local_dt(c, inorm, jnorm, knorm, cfl));
        }
        return smallest_dt;
    } // end estimate_allowed_dt()

    __host__
    void encodeConserved(int level)
    {
        for (auto i=0; i < nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities& U = Q[level*nActiveCells + i];
            c.encode_conserved(U);
        }
        return;
    }

    __host__
    int decodeConserved(int level)
    {
        int bad_cell_count = 0;
        for (auto i=0; i < nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities U = Q[level*nActiveCells + i];
            int flag = c.decode_conserved(U);
            if (flag) { bad_cell_count += 1; }
        }
        return bad_cell_count;
    }

    __host__
    void calculate_fluxes(int x_order)
    {
        for (auto& face : iFaces) {
            FlowState fsL = cells[face.left_cells[0]].fs;
            FlowState fsR = cells[face.right_cells[0]].fs;
            ausmdv(face, fsL, fsR);
        }
        for (auto& face : jFaces) {
            FlowState fsL = cells[face.left_cells[0]].fs;
            FlowState fsR = cells[face.right_cells[0]].fs;
            ausmdv(face, fsL, fsR);
        }
        for (auto& face : kFaces) {
            FlowState& fsL = cells[face.left_cells[0]].fs;
            FlowState& fsR = cells[face.right_cells[0]].fs;
            ausmdv(face, fsL, fsR);
        }
        return;
    }

    __host__
    void eval_dUdt(FVCell& c, ConservedQuantities& dUdt)
    // These are the spatial (RHS) terms in the semi-discrete governing equations.
    {
        number vol_inv = 1.0/c.volume;
        auto& fim = iFaces[c.face[Face::iminus]];
        auto& fip = iFaces[c.face[Face::iplus]];
        auto& fjm = jFaces[c.face[Face::jminus]];
        auto& fjp = jFaces[c.face[Face::jplus]];
        auto& fkm = kFaces[c.face[Face::kminus]];
        auto& fkp = kFaces[c.face[Face::kplus]];
        //
        for (int i=0; i < CQI::n; i++) {
            // Integrate the fluxes across the interfaces that bound the cell.
            number surface_integral = 0.0;
            surface_integral = fim.area*fim.F[i] - fip.area*fip.F[i]
                + fjm.area*fjm.F[i] - fjp.area*fjp.F[i]
                + fkm.area*fkm.F[i] - fkp.area*fkp.F[i];
            // Then evaluate the derivatives of conserved quantity.
            // Note that conserved quantities are stored per-unit-volume.
            dUdt[i] = vol_inv*surface_integral;
        }
        return;
    } // end eval_dUdt()

    __host__
    int update_stage_1(number dt)
    // Predictor step.
    {
        cout << "Start update_state_1 for block " << endl;
        int bad_cell_count = 0;
        for (auto i=0; i < nActiveCells; i++) {
            FVCell& c = cells[i];
            ConservedQuantities dUdt = dQdt[i];
            eval_dUdt(c, dUdt);
            ConservedQuantities U0 = Q[i];
            ConservedQuantities U1 = Q[nActiveCells + i];
            for (int j=0; j < CQI::n; j++) {
                U1[j] = U0[j] + dt*dUdt[j];
            }
            int flag = c.decode_conserved(U1);
            if (flag) { bad_cell_count += 1; }
        }
        cout << "End update_state_1 for block: bad_cell_count=" << bad_cell_count << endl;
        return bad_cell_count;
    } // end update_stage_1()

    __host__
    void copy_conserved_data(int from_level, int to_level)
    {
        for (auto i=0; i < nActiveCells; i++) {
            ConservedQuantities U_from = Q[from_level*nActiveCells + i];
            ConservedQuantities U_to = Q[to_level*nActiveCells + i];
            for (int j=0; j < CQI::n; j++) {
                U_to[j] = U_from[j];
            }
        }
    } // end copy_conserved_data()

}; // end Block

#endif
