#include "hip/hip_runtime.h"
// cell.cu
// Include file for chicken.
// PJ 2022-09-11

#ifndef CELL_INCLUDED
#define CELL_INCLUDED

#include <string>
#include <sstream>
#include <stdexcept>

#include "number.cu"
#include "vector3.cu"
#include "gas.cu"
#include "flow.cu"
#include "vertex.cu"
#include "face.cu"

using namespace std;

namespace Face {
    // Symbolic names for the faces of the cell and of the block.
    constexpr int iminus = 0;
    constexpr int iplus = 1;
    constexpr int jminus = 2;
    constexpr int jplus = 3;
    constexpr int kminus = 4;
    constexpr int kplus = 5;

    array<string,6> names {"iminus", "iplus", "jminus", "jplus", "kminus", "kplus"};
};

int Face_indx_from_name(string name)
{
    if (name == "iminus") return Face::iminus;
    if (name == "iplus") return Face::iplus;
    if (name == "jminus") return Face::jminus;
    if (name == "jplus") return Face::jplus;
    if (name == "kminus") return Face::kminus;
    if (name == "kplus") return Face::kplus;
    throw runtime_error("Invalid face name: " + name);
}


namespace IOvar {
    // Following the new IO model for Eilmer, we set up the accessor functions
    // for the flow data that is held in the flow data files.
    // These accessor functions are associated with the Cell structure.

    // Keep the following list consistent with the GlobalConfig.iovar_names list
    // in chkn_prep.py and with the symbolic constants just below.
    vector<string> names {"pos.x", "pos.y", "pos.z", "vol",
                              "p", "T", "rho", "e", "a",
                              "vel.x", "vel.y", "vel.z"};

    // We will use these symbols to select the varaible of interest.
    constexpr int posx = 0;
    constexpr int posy = posx + 1;
    constexpr int posz = posy + 1;
    constexpr int vol = posz + 1;
    constexpr int p = vol + 1;
    constexpr int T = p + 1;
    constexpr int rho = T + 1;
    constexpr int e = rho + 1;
    constexpr int a = e + 1;
    constexpr int velx = a + 1;
    constexpr int vely = velx + 1;
    constexpr int velz = vely + 1;
    constexpr int n = velz + 1; // number of symbols that point to the flow variables
}


struct FVCell {
    Vector3 pos; // position of centroid
    number volume;
    number iLength, jLength, kLength; // These lengths are used in the interpolation fns.
    FlowState fs;
    // We will keep connections to the pieces compising the cell as indices
    // into the block's arrays.
    // Although we probably don't need build and keep this data for the structured grid,
    // it simplifies some of the geometry and update code and may ease the use of
    // unstructured grids at a later date.
    array<int,8> vtx{0, 0, 0, 0, 0, 0, 0, 0};
    array<int,6> face{0, 0, 0, 0, 0, 0};

    string toString() {
        ostringstream repr;
        repr << "Cell(pos=" << pos.toString() << ", volume=" << volume;
        repr << ", iLength=" << iLength << ", jLength=" << jLength << ", kLength=" << kLength;
        repr << ", fs=" << fs.toString();
        repr << ", vtx=["; for(auto v : vtx) repr << v << ","; repr << "]";
        repr << ", face=["; for(auto v : face) repr << v << ","; repr << "]";
        repr << ")";
        return repr.str();
    }

    void iovar_set(int i, number val)
    {
        switch (i) {
        case IOvar::posx: pos.x = val; break;
        case IOvar::posy: pos.y = val; break;
        case IOvar::posz: pos.z = val; break;
        case IOvar::vol: volume = val; break;
        case IOvar::p: fs.gas.p = val; break;
        case IOvar::T: fs.gas.T = val; break;
        case IOvar::rho: fs.gas.rho = val; break;
        case IOvar::e: fs.gas.e = val; break;
        case IOvar::a: fs.gas.a = val; break;
        case IOvar::velx: fs.vel.x = val; break;
        case IOvar::vely: fs.vel.y = val; break;
        case IOvar::velz: fs.vel.z = val; break;
        default:
            throw runtime_error("Invalid selection for IOvar: "+to_string(i));
        }
    }

    number iovar_get(int i)
    {
        switch (i) {
        case IOvar::posx: return pos.x;
        case IOvar::posy: return pos.y;
        case IOvar::posz: return pos.z;
        case IOvar::vol: return volume;
        case IOvar::p: return fs.gas.p;
        case IOvar::T: return fs.gas.T;
        case IOvar::rho: return fs.gas.rho;
        case IOvar::e: return fs.gas.e;
        case IOvar::a: return fs.gas.a;
        case IOvar::velx: return fs.vel.x;
        case IOvar::vely: return fs.vel.y;
        case IOvar::velz: return fs.vel.z;
        default:
            throw runtime_error("Invalid selection for IOvar: "+to_string(i));
        }
        // So we never return from here.
    }

    __host__ __device__
    number estimate_local_dt(Vector3 inorm, Vector3 jnorm, Vector3 knorm, number cfl)
    {
        // We assume that the cells are (roughly) hexagonal and work with
        // velocities normal to the faces.
        number isignal = iLength/(fabs(fs.vel.dot(inorm))+fs.gas.a);
        number jsignal = jLength/(fabs(fs.vel.dot(jnorm))+fs.gas.a);
        number ksignal = kLength/(fabs(fs.vel.dot(knorm))+fs.gas.a);
        return cfl * fmin(fmin(isignal,jsignal),ksignal);
    }

    __host__ __device__
    void eval_dUdt(ConservedQuantities& dUdt, FVFace faces[])
    // These are the spatial (RHS) terms in the semi-discrete governing equations.
    {
        number vol_inv = 1.0/volume;
        auto& fim = faces[face[Face::iminus]];
        auto& fip = faces[face[Face::iplus]];
        auto& fjm = faces[face[Face::jminus]];
        auto& fjp = faces[face[Face::jplus]];
        auto& fkm = faces[face[Face::kminus]];
        auto& fkp = faces[face[Face::kplus]];
        //
        for (int i=0; i < CQI::n; i++) {
            // Integrate the fluxes across the interfaces that bound the cell.
            number surface_integral = fim.area*fim.F[i] - fip.area*fip.F[i]
                + fjm.area*fjm.F[i] - fjp.area*fjp.F[i]
                + fkm.area*fkm.F[i] - fkp.area*fkp.F[i];
            // Then evaluate the derivatives of conserved quantity.
            // Note that conserved quantities are stored per-unit-volume.
            dUdt[i] = vol_inv*surface_integral;
        }
        return;
    } // end eval_dUdt()

}; // end Cell

#endif
