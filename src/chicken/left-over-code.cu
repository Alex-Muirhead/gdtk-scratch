#include "hip/hip_runtime.h"
// left-over-code.cu
// CUDA bootcamp left-overs.
// Put here so that code might be cribbed.


//---------------------------------------------------------------------------
//
// Bits left over from the CUDA workshop experiment.
// Initial hack adapts the vector addition example from the CUDA workshop
// to look a bit closer to our Puffin CFD code.
//
void host_process(vector<FlowState>& fss)
{
    for (auto& fs : fss) {
        auto& gas = fs.gas;
        auto& vel = fs.vel;
        number v2 = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
        number v = sqrt(v2);
        number M = v/gas.a;
        number g = GasModel::g;
        number t1 = 1.0f + 0.5f*(g-1.0)*M*M;
        // Compute stagnation condition.
        number p_total = gas.p * pow(t1, (g/(g-1.0)));
        number T_total = gas.T * t1;
        gas.p = p_total;
        gas.T = T_total;
        gas.update_from_pT();
        vel = {0.0, 0.0, 0.0};
    }
    cout << "inside host_process: fss[0]= " << fss[0].toString() << endl;
}

__global__ void device_process(FlowState* fss, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    //
    if (idx < N) {
        auto& fs = fss[idx];
        auto gas = fs.gas;
        auto vel = fs.vel;
        number v2 = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
        number v = sqrt(v2);
        number M = v/gas.a;
        number g = GasModel::g;
        number t1 = 1.0f + 0.5f*(g-1.0f)*M*M;
        // Compute stagnation condition.
        number p_total = gas.p * pow(t1, (g/(g-1.0f)));
        number T_total = gas.T * t1;
        gas.p = p_total;
        gas.T = T_total;
        gas.update_from_pT();
        vel = {0.0f, 0.0f, 0.0f};
        fs.gas = gas;
        fs.vel = vel;
    }
}

void print_sample(vector<FlowState> fss)
{
    for (int idx=0; idx < 3; idx++) {
        auto& fs = fss[idx];
        cout << "fs= " << fs.toString() << endl;
    }
    cout << "..." << endl;
    int N = fss.size();
    for (int idx=N-3; idx < N; idx++) {
        auto& fs = fss[idx];
        cout << "fs=" << fs.toString() << endl;
    }
}

void do_something()
{
    // Host data is in a standard C++ vector.
    vector<FlowState> fss_h;
    const int N = 32*512;
    for (int idx=0; idx < N; idx++) {
        auto gas = GasState{0.0, 0.0, 100.0e3, 300.0, 0.0};
        gas.update_from_pT();
        auto vel = Vector3{1000.0, 99.0, 0.0};
        fss_h.push_back(FlowState{gas, vel});
    }
    #ifdef CUDA
    if (!filesystem::exists(filesystem::status("/proc/driver/nvidia"))) {
        throw runtime_error("Cannot find NVIDIA driver in /proc/driver.");
    }
    int nDevices;
    hipGetDeviceCount(&nDevices);
    cout << "Found " << nDevices << " CUDA devices." << endl;
    if (nDevices > 0) {
        cout << "We have a CUDA device, so use it." << endl;
        // Pointer to device arrays.
        FlowState* fss_d;
        int sze = N * sizeof(FlowState);
        hipMalloc(&fss_d, sze);
        hipMemcpy(fss_d, fss_h.data(), sze, hipMemcpyHostToDevice);
        //
        const int threads_per_block = 128;
        const int nblocks = N/threads_per_block;
        device_process<<<nblocks,threads_per_block>>>(fss_d, N);
        cout << hipGetErrorString(hipGetLastError()) << endl;
        //
        hipMemcpy(fss_h.data(), fss_d, sze, hipMemcpyDeviceToHost);
        hipFree(fss_d);
    } else {
        cout << "Fall back to CPU-only processing." << endl;
        host_process(fss_h);
    }
    #else
    host_process(fss_h);
    #endif
    print_sample(fss_h);
    fss_h.resize(0);
    return;
}
